#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>
#include <c10/cuda/CUDAGuard.h>

#include <algorithm>
#include <chrono>

#include "detail/ops/Ops.h"
#include "detail/ops/convolution/backend/ConvOps.h"

namespace fvdb {
namespace detail {
namespace ops {

template <typename scalar_t>
__global__ void gatherKernel(const int n_k, const int n_in, const int c,
                              const scalar_t *in_feat, scalar_t *out_feat,
                              const int *kmap, const bool transpose) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int i = index / c;
    int j = index % c;
    if (i >= n_k) return;
    int in_pos = kmap[2 * i + transpose];
    if (in_pos < 0) return;
    out_feat[i * c + j] = in_feat[in_pos * c + j];
}

template <typename scalar_t>
__global__ void scatterKernel(const int n_in, const int n_out, const int c,
                               const scalar_t *in_feat, scalar_t *out_feat,
                               const int *kmap, const bool transpose) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int i = index / c;
    int j = index % c;
    if (i >= n_in) return;
    int out_pos = kmap[2 * i + 1 - transpose];
    if (out_pos < 0) return;
    out_feat[out_pos * c + j] += in_feat[i * c + j];
}

template <typename scalar_t>
static void scatterCpu(const int n_in, const int n_out, const int c,
                 const scalar_t *in_feat, scalar_t *out_feat, const int *kmap,
                 const bool transpose) {
    for (int i = 0; i < n_in; i++) {
        int out_pos = kmap[2 * i + 1 - transpose];
        if (out_pos < 0) {
            continue;
        }
        #pragma omp parallel for
        for (int j = 0; j < c; j++) {
            out_feat[out_pos * c + j] += in_feat[i * c + j];
        }
    }
}

template <typename scalar_t>
static void gatherCpu(const int n_k, const int n_in, const int c,
                const scalar_t *in_feat, scalar_t *out_feat, const int *kmap,
                const bool transpose) {
    for (int i = 0; i < n_k; i++) {
        int in_pos = kmap[2 * i + transpose];
        if (in_pos < 0) {
            continue;
        }
#pragma omp parallel for
        for (int j = 0; j < c; j++) {
            out_feat[i * c + j] = in_feat[in_pos * c + j];
        }
    }
}


// in_feat: (N, c) N=# of input points, c = input channels
// out_feat: (M, o) M=# of output points, o = output channels
//                  for stride=1, M=N. For stride>1, the N input coords
//                  are requantized to M points with grid size (stride *
//                  cur_stride)
// kernel: (k^3, c, o) for a 3D convolution of length k
// neighbor_map: (a, 2) the hash table query results from out_coords to
// in_coords
//                      where neighbor_map[:,0] is the index of the output
//                      feature and neighbor_map[:,1] is the index of the input
//                      feature
// neighbor_offset: (k^3) count of active weights based on neighbor_map
//                      with unused weights having 0 and neighbor_offset[k^3/2]
//                      holding w[0,0].
template <>
void dispatchSparseConvolutionKernelMap<torch::kCUDA>(at::Tensor in_feat, at::Tensor out_feat,
                                                    at::Tensor kernel, at::Tensor neighbor_map,
                                                    at::Tensor neighbor_offset,
                                                    const bool transpose,
                                                    const bool middleAcceleration) {
    TORCH_CHECK(in_feat.device().is_cuda(), "in_feat must be a CUDA tensor");
    TORCH_CHECK(in_feat.device().has_index(), "in_feat must have a device index");
    TORCH_CHECK(in_feat.device() == out_feat.device(), "All tensors must be on the same device, got in_feat.device() = ",
                in_feat.device(), ", out_feat.device() = ", out_feat.device());
    TORCH_CHECK(in_feat.device() == kernel.device(), "All tensors must be on the same device, got in_feat.device() = ",
                in_feat.device(), ", kernel.device() = ", kernel.device());
    TORCH_CHECK(in_feat.device() == neighbor_map.device(), "All tensors must be on the same device, got in_feat.device() = ",
                in_feat.device(), ", neighbor_map.device() = ", neighbor_map.device());
    TORCH_CHECK(neighbor_offset.device().is_cpu(), "neighborhood_offset must be on the CPU because torch_sparse conv is wack");

    c10::cuda::CUDAGuard deviceGuard(in_feat.device());

    if (in_feat.size(1) != kernel.size(1)) {
        throw std::invalid_argument("Input feature size and kernel size mismatch");
    }

    bool is_half = in_feat.scalar_type() == at::ScalarType::Half;

    int n_in_feats = in_feat.size(0);
    int n_in_channels = in_feat.size(1);
    int n_out_feats = out_feat.size(0);
    int n_out_channels = out_feat.size(1);

    int kernel_volume = kernel.size(0);

    // memory optimization
    bool precompute_mid = false;
    int mid_kernel = kernel_volume / 2;
    int in_buffer_size = 1;
    // we can precompute features for w[0,0] which avoids gather/scatter
    if (kernel_volume % 2 == 1 && n_in_feats == n_out_feats && middleAcceleration) {
        precompute_mid = true;
        in_buffer_size =
                *std::max_element(neighbor_offset.data_ptr<int>(),
                                  neighbor_offset.data_ptr<int>() + mid_kernel);
        in_buffer_size = std::max(
                in_buffer_size,
                *std::max_element(neighbor_offset.data_ptr<int>() + mid_kernel + 1,
                                  neighbor_offset.data_ptr<int>() + kernel_volume));
        in_buffer_size = std::max(in_buffer_size, 1);

        // (N, c) X (c, o) = (N, o)
        torch::mm_out(out_feat, in_feat, kernel[mid_kernel]);
    } else {
        in_buffer_size =
                *std::max_element(neighbor_offset.data_ptr<int>(),
                                  neighbor_offset.data_ptr<int>() + kernel_volume);
    }

    auto options =
            torch::TensorOptions().dtype(in_feat.dtype()).device(in_feat.device());
    auto in_buffer = torch::zeros({in_buffer_size, n_in_channels}, options);
    auto out_buffer = torch::zeros({in_buffer_size, n_out_channels}, options);
    int cur_offset = 0;
    // gather/gemm/scatter on each weight
    for (int i = 0; i < kernel_volume; i++) {
        int n_active_feats = neighbor_offset.data_ptr<int>()[i];
        // if there's no active features for this weight, skip it
        if (n_active_feats == 0) {
            continue;
        }

        // if w[0,0] was precomputed above, skip it
        if ((i == mid_kernel) && precompute_mid) {
            cur_offset += 2 * n_active_feats;
            continue;
        }

        // in_buffer_activated (i, c) holds the dense input features from gather
        // for i = n_active_feats (# of features in the activated kernel from
        // neighbor_offset) out_buffer_activated (i, o) holds the dense output
        // features to scatter
        at::Tensor out_buffer_activated;
        at::Tensor in_buffer_activated;
        if (is_half) {
            out_buffer_activated =
                    torch::from_blob(out_buffer.data_ptr<at::Half>(),
                                     {n_active_feats, n_out_channels}, options);
            in_buffer_activated =
                    torch::from_blob(in_buffer.data_ptr<at::Half>(),
                                     {n_active_feats, n_in_channels}, options);
        } else {
            out_buffer_activated =
                    torch::from_blob(out_buffer.data_ptr(),
                                     {n_active_feats, n_out_channels}, options);
            in_buffer_activated =
                    torch::from_blob(in_buffer.data_ptr(),
                                     {n_active_feats, n_in_channels}, options);
        }

        // gather n_active_feats dense features from N sparse input features with c
        // feature dimensions
        AT_DISPATCH_FLOATING_TYPES_AND_HALF(
                in_feat.scalar_type(), "convolution_forward_cuda", ([&] {
            gatherKernel<scalar_t>
            <<<ceil((double)(n_active_feats * n_in_channels) / 256), 256>>>(
                    n_active_feats, n_in_feats, n_in_channels,
                    in_feat.data_ptr<scalar_t>(),
                    in_buffer_activated.data_ptr<scalar_t>(),
                    neighbor_map.data_ptr<int>() + cur_offset, transpose);
        }));

        // gemm: (i, c) X (c, o) = (i, o)
        torch::mm_out(out_buffer_activated, in_buffer_activated, kernel[i]);

        // scatter n_active_feats dense features into n_out_feats output features of
        // dimension n_out_channels
        AT_DISPATCH_FLOATING_TYPES_AND_HALF(
                in_feat.scalar_type(), "convolution_forward_cuda", ([&] {
            scatterKernel<scalar_t>
            <<<ceil((double)(n_active_feats * n_out_channels) / 256), 256>>>(
                    n_active_feats, n_out_feats, n_out_channels,
                    out_buffer_activated.data_ptr<scalar_t>(),
                    out_feat.data_ptr<scalar_t>(),
                    neighbor_map.data_ptr<int>() + cur_offset, transpose);
        }));

        cur_offset += 2 * n_active_feats;
    }
}

template <>
void dispatchSparseConvolutionKernelMapGrad<torch::kCUDA>(at::Tensor in_feat, at::Tensor grad_in_feat,
                                                        at::Tensor grad_out_feat, at::Tensor kernel,
                                                        at::Tensor grad_kernel, at::Tensor neighbor_map,
                                                        at::Tensor neighbor_offset,
                                                        const bool transpose) {
    TORCH_CHECK(in_feat.device().is_cuda(), "in_feat must be a CUDA tensor");
    TORCH_CHECK(in_feat.device().has_index(), "in_feat must have a device index");
    TORCH_CHECK(in_feat.device() == grad_in_feat.device(), "All tensors must be on the same device");
    TORCH_CHECK(in_feat.device() == grad_out_feat.device(), "All tensors must be on the same device");
    TORCH_CHECK(in_feat.device() == kernel.device(), "All tensors must be on the same device");
    TORCH_CHECK(in_feat.device() == grad_kernel.device(), "All tensors must be on the same device");
    TORCH_CHECK(in_feat.device() == neighbor_map.device(), "All tensors must be on the same device");
    TORCH_CHECK(neighbor_offset.device().is_cpu(), "neighborhood_offset must be on the CPU because torch_sparse conv is wack");

    c10::cuda::CUDAGuard deviceGuard(in_feat.device());

    grad_in_feat.resize_as_(in_feat);
    grad_in_feat.zero_();
    grad_kernel.resize_as_(kernel);
    grad_kernel.zero_();

    bool is_half = in_feat.scalar_type() == at::ScalarType::Half;
    int n_in_feats = in_feat.size(0);
    int n_in_channels = in_feat.size(1);
    int n_out_feats = grad_out_feat.size(0);
    int n_out_channels = kernel.size(-1);

    int kernel_volume = kernel.size(0);
    bool flag = false;
    int in_buffer_size;
    in_buffer_size =
            *std::max_element(neighbor_offset.data_ptr<int>(),
                              neighbor_offset.data_ptr<int>() + kernel_volume);

    auto options =
            torch::TensorOptions().dtype(in_feat.dtype()).device(in_feat.device());
    auto in_buffer = torch::zeros({in_buffer_size, in_feat.size(1)}, options);
    auto in_grad_buffer =
            torch::zeros({in_buffer_size, in_feat.size(1)}, options);
    auto out_grad_buffer =
            torch::zeros({in_buffer_size, kernel.size(2)}, options);

    int cur_offset = 0;
    for (int i = 0; i < kernel_volume; i++) {
        auto kernel_grad_buffer = grad_kernel[i];
        int n_active_feats = neighbor_offset.data_ptr<int>()[i];
        if (flag && (i == kernel_volume / 2)) {
            cur_offset += 2 * n_active_feats;
            continue;
        }

        if (n_active_feats == 0) {
            continue;
        }

        // Can't figure out a cleaner way to do this
        at::Tensor out_grad_buffer_activated;
        at::Tensor in_grad_buffer_activated;
        at::Tensor in_buffer_activated;
        if (is_half) {
            out_grad_buffer_activated =
                    torch::from_blob(out_grad_buffer.data_ptr<at::Half>(),
                                     {n_active_feats, kernel.size(2)}, options);
            in_grad_buffer_activated =
                    torch::from_blob(in_grad_buffer.data_ptr<at::Half>(),
                                     {n_active_feats, in_feat.size(1)}, options);
            in_buffer_activated =
                    torch::from_blob(in_buffer.data_ptr<at::Half>(),
                                     {n_active_feats, in_feat.size(1)}, options);
        } else {
            out_grad_buffer_activated =
                    torch::from_blob(out_grad_buffer.data_ptr(),
                                     {n_active_feats, kernel.size(2)}, options);
            in_grad_buffer_activated =
                    torch::from_blob(in_grad_buffer.data_ptr(),
                                     {n_active_feats, in_feat.size(1)}, options);
            in_buffer_activated =
                    torch::from_blob(in_buffer.data_ptr(),
                                     {n_active_feats, in_feat.size(1)}, options);
        }

        // gather
        AT_DISPATCH_FLOATING_TYPES_AND_HALF(
                in_feat.scalar_type(), "convolution_forward_cuda", ([&] {
            gatherKernel<scalar_t>
            <<<ceil((double)(n_active_feats * n_out_channels) / 256), 256>>>(
                    n_active_feats, n_out_feats, n_out_channels,
                    grad_out_feat.data_ptr<scalar_t>(),
                    out_grad_buffer_activated.data_ptr<scalar_t>(),
                    neighbor_map.data_ptr<int>() + cur_offset, !transpose);
        }));

        AT_DISPATCH_FLOATING_TYPES_AND_HALF(
                in_feat.scalar_type(), "convolution_forward_cuda", ([&] {
            gatherKernel<scalar_t>
            <<<ceil((double)(n_active_feats * n_in_channels) / 256), 256>>>(
                    n_active_feats, n_in_feats, n_in_channels,
                    in_feat.data_ptr<scalar_t>(),
                    in_buffer_activated.data_ptr<scalar_t>(),
                    neighbor_map.data_ptr<int>() + cur_offset, transpose);
        }));

        // gemm
        torch::mm_out(in_grad_buffer_activated, out_grad_buffer_activated,
                      torch::transpose(kernel[i], 0, 1));
        torch::mm_out(kernel_grad_buffer,
                      torch::transpose(in_buffer_activated, 0, 1),
                      out_grad_buffer_activated);

        // scatter
        AT_DISPATCH_FLOATING_TYPES_AND_HALF(
                in_feat.scalar_type(), "convolution_forward_cuda", ([&] {
            scatterKernel<scalar_t>
            <<<ceil((double)(n_active_feats * n_in_channels) / 256), 256>>>(
                    n_active_feats, n_in_feats, n_in_channels,
                    in_grad_buffer_activated.data_ptr<scalar_t>(),
                    grad_in_feat.data_ptr<scalar_t>(),
                    neighbor_map.data_ptr<int>() + cur_offset, !transpose);
        }));

        cur_offset += 2 * n_active_feats;
    }
}




template <>
void dispatchSparseConvolutionKernelMap<torch::kCPU>(torch::Tensor in_feat, torch::Tensor out_feat,
                                                   torch::Tensor kernel, torch::Tensor neighbor_map,
                                                   torch::Tensor neighbor_offset,
                                                   bool transpose,
                                                   bool middleAcceleration) {
    if (in_feat.size(1) != kernel.size(1)) {
        throw std::invalid_argument("Input feature size and kernel size mismatch");
    }

    int out_nrows = out_feat.size(0);
    out_feat.resize_({out_nrows, kernel.size(2)});
    out_feat.zero_();

    int kernel_volume = kernel.size(0);
    int in_buffer_size = 1;
    bool flag = false;
    // memory optimization
    if (kernel_volume % 2 && out_nrows == in_feat.size(0) && middleAcceleration) {
        flag = true;
        in_buffer_size =
                *std::max_element(neighbor_offset.data_ptr<int>(),
                                  neighbor_offset.data_ptr<int>() + kernel_volume / 2);
        in_buffer_size =
                std::max(in_buffer_size,
                         *std::max_element(
                                 neighbor_offset.data_ptr<int>() + kernel_volume / 2 + 1,
                                 neighbor_offset.data_ptr<int>() + kernel_volume));
        in_buffer_size = std::max(in_buffer_size, 1);

        torch::mm_out(out_feat, in_feat, kernel[kernel_volume / 2]);
    } else {
        in_buffer_size =
                *std::max_element(neighbor_offset.data_ptr<int>(),
                                  neighbor_offset.data_ptr<int>() + kernel_volume);
    }

    auto options =
            torch::TensorOptions().dtype(in_feat.dtype()).device(in_feat.device());
    auto in_buffer = torch::zeros({in_buffer_size, in_feat.size(1)}, options);
    auto out_buffer = torch::zeros({in_buffer_size, kernel.size(2)}, options);
    int cur_offset = 0;
    for (int i = 0; i < kernel_volume; i++) {
        if (flag && (i == kernel_volume / 2)) {
            cur_offset += 2 * neighbor_offset.data_ptr<int>()[i];
            continue;
        }

        if (neighbor_offset.data_ptr<int>()[i] == 0) {
            continue;
        }

        auto out_buffer_activated = torch::from_blob(
                out_buffer.data_ptr(),
                {neighbor_offset.data_ptr<int>()[i], kernel.size(2)}, options);
        auto in_buffer_activated = torch::from_blob(
                in_buffer.data_ptr(),
                {neighbor_offset.data_ptr<int>()[i], in_feat.size(1)}, options);

        // gather
        AT_DISPATCH_FLOATING_TYPES(in_feat.scalar_type(), "gatherCpu", [&]() {
            gatherCpu(in_buffer_activated.size(0), in_feat.size(0), kernel.size(1),
                       in_feat.data_ptr<scalar_t>(), in_buffer_activated.data_ptr<scalar_t>(),
                       neighbor_map.data_ptr<int>() + cur_offset, transpose);
        });

        // matmul
        torch::mm_out(out_buffer_activated, in_buffer_activated, kernel[i]);

        // scatter
        AT_DISPATCH_FLOATING_TYPES(out_feat.scalar_type(), "scatterCpu", [&](){
            scatterCpu(neighbor_offset.data_ptr<int>()[i], out_nrows, kernel.size(2),
                        out_buffer_activated.data_ptr<scalar_t>(),
                        out_feat.data_ptr<scalar_t>(),
                        neighbor_map.data_ptr<int>() + cur_offset, transpose);
        });
        cur_offset += 2 * neighbor_offset.data_ptr<int>()[i];
    }
}


template <>
void dispatchSparseConvolutionKernelMapGrad<torch::kCPU>(at::Tensor in_feat, at::Tensor grad_in_feat,
                                                       at::Tensor grad_out_feat, at::Tensor kernel,
                                                       at::Tensor grad_kernel, at::Tensor neighbor_map,
                                                       at::Tensor neighbor_offset,
                                                       bool transpose) {
    grad_in_feat.resize_as_(in_feat);
    grad_in_feat.zero_();
    grad_kernel.resize_as_(kernel);
    grad_kernel.zero_();

    int kernel_volume = kernel.size(0);
    bool flag = false;
    int in_buffer_size;
    in_buffer_size =
            *std::max_element(neighbor_offset.data_ptr<int>(),
                              neighbor_offset.data_ptr<int>() + kernel_volume);

    auto options =
            torch::TensorOptions().dtype(in_feat.dtype()).device(in_feat.device());
    auto in_buffer = torch::zeros({in_buffer_size, in_feat.size(1)}, options);
    auto in_grad_buffer =
            torch::zeros({in_buffer_size, in_feat.size(1)}, options);
    auto out_grad_buffer =
            torch::zeros({in_buffer_size, kernel.size(2)}, options);

    int cur_offset = 0;
    for (int i = 0; i < kernel_volume; i++) {
        auto kernel_grad_buffer = grad_kernel[i];
        if (flag && (i == kernel_volume / 2)) {
            cur_offset += 2 * neighbor_offset.data_ptr<int>()[i];
            continue;
        }

        if (neighbor_offset.data_ptr<int>()[i] == 0) {
            continue;
        }

        auto out_grad_buffer_activated = torch::from_blob(
                out_grad_buffer.data_ptr(),
                {neighbor_offset.data_ptr<int>()[i], kernel.size(2)}, options);
        auto in_grad_buffer_activated = torch::from_blob(
                in_grad_buffer.data_ptr(),
                {neighbor_offset.data_ptr<int>()[i], in_feat.size(1)}, options);
        auto in_buffer_activated = torch::from_blob(
                in_buffer.data_ptr(),
                {neighbor_offset.data_ptr<int>()[i], in_feat.size(1)}, options);

        // gather
        AT_DISPATCH_FLOATING_TYPES(grad_out_feat.scalar_type(), "gatherCpu", [&](){
            gatherCpu(out_grad_buffer_activated.size(0), grad_out_feat.size(0),
                       kernel.size(2), grad_out_feat.data_ptr<scalar_t>(),
                       out_grad_buffer_activated.data_ptr<scalar_t>(),
                       neighbor_map.data_ptr<int>() + cur_offset, !transpose);
        });
        AT_DISPATCH_FLOATING_TYPES(grad_out_feat.scalar_type(), "gatherCpu", [&](){
            gatherCpu(in_buffer_activated.size(0), in_feat.size(0), kernel.size(1),
                       in_feat.data_ptr<scalar_t>(), in_buffer_activated.data_ptr<scalar_t>(),
                       neighbor_map.data_ptr<int>() + cur_offset, transpose);
        });

        // matmul
        torch::mm_out(in_grad_buffer_activated, out_grad_buffer_activated,
                      torch::transpose(kernel[i], 0, 1));
        torch::mm_out(kernel_grad_buffer,
                      torch::transpose(in_buffer_activated, 0, 1),
                      out_grad_buffer_activated);

        // scatter
        AT_DISPATCH_FLOATING_TYPES(grad_out_feat.scalar_type(), "scatterCpu", [&](){
            scatterCpu(neighbor_offset.data_ptr<int>()[i], in_feat.size(0),
                        kernel.size(1), in_grad_buffer_activated.data_ptr<scalar_t>(),
                        grad_in_feat.data_ptr<scalar_t>(),
                        neighbor_map.data_ptr<int>() + cur_offset, !transpose);
        });

        cur_offset += 2 * neighbor_offset.data_ptr<int>()[i];
    }
}


} // namespace ops
} // namespace detail
} // namespace fvdb