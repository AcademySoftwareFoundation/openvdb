#include "hip/hip_runtime.h"
// Copyright Contributors to the OpenVDB Project
// SPDX-License-Identifier: Apache-2.0
//
#include "nanovdb/NanoVDB.h"

#include <detail/utils/AccessorHelpers.cuh>
#include <detail/utils/cuda/ForEachCUDA.cuh>

#include <THC/THCAtomics.cuh>
#include <c10/cuda/CUDAException.h>

namespace fvdb {
namespace detail {
namespace ops {

template <typename ScalarType>
__hostdev__ inline void
readIntoDenseVoxelCallback(
    int32_t batchIdx,
    int32_t leafIdx,
    int32_t voxelIdx,
    int32_t channelIdx,
    GridBatchImpl::Accessor<nanovdb::ValueOnIndex> batchHandle,
    torch::PackedTensorAccessor64<int32_t, 2, torch::RestrictPtrTraits> denseOrigins, // [B, 3]
    torch::PackedTensorAccessor64<ScalarType, 2, torch::RestrictPtrTraits>
        inSparseTensor,                                                               // [B*N, C]
    torch::PackedTensorAccessor64<ScalarType, 5, torch::RestrictPtrTraits> outDenseTensor) {
    using LeafNodeT = typename nanovdb::OnIndexGrid::LeafNodeType;

    const nanovdb::OnIndexGrid *gpuGrid = batchHandle.grid(batchIdx);
    const nanovdb::Coord denseDim(
        outDenseTensor.size(1), outDenseTensor.size(2), outDenseTensor.size(3));
    const nanovdb::Coord denseOrigin(
        denseOrigins[batchIdx][0], denseOrigins[batchIdx][1], denseOrigins[batchIdx][2]);
    const nanovdb::CoordBBox bbox(denseOrigin, denseOrigin + denseDim - nanovdb::Coord(1, 1, 1));
    const int64_t baseOffset = batchHandle.voxelOffset(batchIdx);

    const LeafNodeT &leaf       = gpuGrid->tree().template getFirstNode<0>()[leafIdx];
    const nanovdb::Coord voxIjk = leaf.offsetToGlobalCoord(voxelIdx);

    const bool isActive = leaf.isActive(voxelIdx);

    const nanovdb::Coord ijk = voxIjk - denseOrigin;
    const int64_t offset     = baseOffset + leaf.getValue(voxelIdx) - 1;

    if (isActive && bbox.isInside(voxIjk)) {
        outDenseTensor[batchIdx][ijk[0]][ijk[1]][ijk[2]][channelIdx] =
            inSparseTensor[offset][channelIdx];
    }
}

template <typename ScalarType>
void
readIntoDenseCPU(const GridBatchImpl::Accessor<nanovdb::ValueOnIndex> &gridHandle,
                 const torch::TensorAccessor<ScalarType, 2> inGridData,
                 const torch::TensorAccessor<int32_t, 2> denseOrigins,
                 torch::TensorAccessor<ScalarType, 5> outDenseTensor,
                 bool isContiguous) {
    for (size_t bi = 0; bi < gridHandle.batchSize(); bi += 1) {
        const nanovdb::OnIndexGrid *grid = gridHandle.grid(bi);

        const nanovdb::Coord bbmin(denseOrigins[bi][0], denseOrigins[bi][1], denseOrigins[bi][2]);
        const nanovdb::Coord bbsize(
            outDenseTensor.size(1), outDenseTensor.size(2), outDenseTensor.size(3));
        const nanovdb::CoordBBox bbox(bbmin, bbmin + bbsize - nanovdb::Coord(1, 1, 1));
        const int64_t baseOffset = gridHandle.voxelOffset(bi);

        auto outBatch = outDenseTensor[bi];

        for (auto it = ActiveVoxelIterator<-1>(grid->tree(), baseOffset); it.isValid(); it++) {
            const nanovdb::Coord voxIjk = it->first;
            if (bbox.isInside(voxIjk)) {
                const nanovdb::Coord ijk = voxIjk - bbox.min();

                if (isContiguous) {
                    memcpy(outBatch[ijk[0]][ijk[1]][ijk[2]].data(),
                           inGridData[it->second].data(),
                           inGridData.size(1) * sizeof(ScalarType));
                } else {
                    for (int c = 0; c < inGridData.size(1); ++c) {
                        outBatch[ijk[0]][ijk[1]][ijk[2]][c] = inGridData[it->second][c];
                    }
                }
            }
        }
    }
}

template <>
void
dispatchReadIntoDense<torch::kCUDA>(const GridBatchImpl &batchHdl,
                                    const torch::Tensor &inGridData,
                                    const torch::Tensor &denseOrigins,
                                    torch::Tensor &outDenseTensor) {
    AT_DISPATCH_V2(
        outDenseTensor.scalar_type(),
        "readIntoDense",
        AT_WRAP([&]() {
            auto outDenseAcc =
                outDenseTensor.packed_accessor64<scalar_t, 5, torch::RestrictPtrTraits>();
            auto denseOriginsAcc =
                denseOrigins.packed_accessor64<int32_t, 2, torch::RestrictPtrTraits>();
            auto inGridDataAcc =
                inGridData.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>();
            auto callback = [=] __device__(
                                int32_t bidx,
                                int32_t lidx,
                                int32_t vidx,
                                int32_t cidx,
                                GridBatchImpl::Accessor<nanovdb::ValueOnIndex> batchAcc) {
                readIntoDenseVoxelCallback<scalar_t>(
                    bidx, lidx, vidx, cidx, batchAcc, denseOriginsAcc, inGridDataAcc, outDenseAcc);
            };
            forEachVoxelCUDA<nanovdb::ValueOnIndex>(1024, inGridData.size(1), batchHdl, callback);
        }),
        AT_EXPAND(AT_FLOATING_TYPES),
        c10::kHalf,
        c10::kBFloat16);
}

template <>
void
dispatchReadIntoDense<torch::kCPU>(const GridBatchImpl &gridHdl,
                                   const torch::Tensor &inGridData,
                                   const torch::Tensor &denseOrigins,
                                   torch::Tensor &outDenseTensor) {
    bool isContiguous = inGridData.is_contiguous() && outDenseTensor.is_contiguous();

    AT_DISPATCH_V2(outDenseTensor.scalar_type(),
                   "readIntoDense",
                   AT_WRAP([&]() {
                       readIntoDenseCPU(gridHdl.hostAccessor<nanovdb::ValueOnIndex>(),
                                        inGridData.accessor<scalar_t, 2>(),
                                        denseOrigins.accessor<int32_t, 2>(),
                                        outDenseTensor.accessor<scalar_t, 5>(),
                                        isContiguous);
                   }),
                   AT_EXPAND(AT_FLOATING_TYPES),
                   c10::kHalf,
                   c10::kBFloat16);
}

} // namespace ops
} // namespace detail
} // namespace fvdb
