#include "hip/hip_runtime.h"
// Copyright Contributors to the OpenVDB Project
// SPDX-License-Identifier: Apache-2.0
//
#include "GsplatTypes.cuh"
#include "VectorTypes.cuh"
#include <detail/ops/Ops.h>

#include <ATen/cuda/Atomic.cuh>

#include <hip/hip_cooperative_groups.h>

#include <hipcub/hipcub.hpp>

namespace fvdb {
namespace detail {
namespace ops {

namespace cg = cooperative_groups;

/****************************************************************************
 * Rasterization to Pixels Backward Pass
 ****************************************************************************/

template <uint32_t COLOR_DIM, uint32_t N_OUTER_DIMS, typename S>
__global__ void
rasterize_to_pixels_bwd_kernel(
    const uint32_t C, const uint32_t N, const uint32_t n_isects, const bool packed,
    // fwd inputs
    const vec2<S> *__restrict__ means2d, // [C, N, 2] or [nnz, 2]
    const vec3<S> *__restrict__ conics,  // [C, N, 3] or [nnz, 3]
    // const S *__restrict__ colors,        // [C, N, COLOR_DIM] or [nnz, COLOR_DIM]
    torch::PackedTensorAccessor64<S, N_OUTER_DIMS + 1, torch::RestrictPtrTraits>
        colors,                        // [C, N, COLOR_DIM] or [nnz, COLOR_DIM]
    const S *__restrict__ opacities,   // [C, N] or [nnz]
    const S *__restrict__ backgrounds, // [C, COLOR_DIM] or [nnz, COLOR_DIM]
    const bool *__restrict__ masks,    // [C, tile_height, tile_width]
    const uint32_t image_width, const uint32_t image_height, const uint32_t image_origin_w,
    const uint32_t image_origin_h, const uint32_t tile_origin_w, const uint32_t tile_origin_h,
    const uint32_t tile_size, const uint32_t tile_width, const uint32_t tile_height,
    const int32_t *__restrict__ tile_offsets, // [C, tile_height, tile_width]
    const int32_t *__restrict__ flatten_ids,  // [n_isects]
    // fwd outputs
    const S *__restrict__ render_alphas,  // [C, image_height, image_width, 1]
    const int32_t *__restrict__ last_ids, // [C, image_height, image_width]
    // grad outputs
    const S *__restrict__ v_render_colors, // [C, image_height, image_width,
                                           // COLOR_DIM]
    const S *__restrict__ v_render_alphas, // [C, image_height, image_width, 1]
    // grad inputs
    vec2<S> *__restrict__ v_means2d_abs, // [C, N, 2] or [nnz, 2]
    vec2<S> *__restrict__ v_means2d,     // [C, N, 2] or [nnz, 2]
    vec3<S> *__restrict__ v_conics,      // [C, N, 3] or [nnz, 3]
    S *__restrict__ v_colors,            // [C, N, COLOR_DIM] or [nnz, COLOR_DIM]
    S *__restrict__ v_opacities          // [C, N] or [nnz]
) {
    auto     block     = cg::this_thread_block();
    uint32_t camera_id = block.group_index().x;

    // blockIdx runs from [0, num_tiles_h] x [0, num_tiles_w]
    const int32_t tile_id = (block.group_index().y + tile_origin_h) * tile_width +
                            block.group_index().z + tile_origin_w;
    // Pixel coordinates run from [0, height] x [0, width]
    const uint32_t i = block.group_index().y * tile_size + block.thread_index().y;
    const uint32_t j = block.group_index().z * tile_size + block.thread_index().x;

    tile_offsets += camera_id * tile_height * tile_width;
    render_alphas += camera_id * image_height * image_width;
    last_ids += camera_id * image_height * image_width;
    v_render_colors += camera_id * image_height * image_width * COLOR_DIM;
    v_render_alphas += camera_id * image_height * image_width;
    if (backgrounds != nullptr) {
        backgrounds += camera_id * COLOR_DIM;
    }
    if (masks != nullptr) {
        masks += camera_id * tile_height * tile_width;
    }

    // when the mask is provided, do nothing and return if
    // this tile is labeled as False
    if (masks != nullptr && !masks[tile_id]) {
        return;
    }

    const S px = (S)(j + image_origin_w) + 0.5f;
    const S py = (S)(i + image_origin_h) + 0.5f;

    // clamp this value to the last pixel
    const int32_t pix_id = min(i * image_width + j, image_width * image_height - 1);

    // keep not rasterizing threads around for reading data
    const bool inside = (i < image_height && j < image_width);

    // have all threads in tile process the same gaussians in batches
    // first collect gaussians between range.x and range.y in batches
    // which gaussians to look through in this tile
    int32_t        range_start = tile_offsets[tile_id];
    int32_t        range_end   = (camera_id == C - 1) && (tile_id == tile_width * tile_height - 1)
                                     ? n_isects
                                     : tile_offsets[tile_id + 1];
    const uint32_t block_size  = block.size();
    const uint32_t num_batches = (range_end - range_start + block_size - 1) / block_size;

    extern __shared__ int s[];
    int32_t              *id_batch = (int32_t *)s;                      // [block_size]
    vec3<S>              *xy_opacity_batch =
        reinterpret_cast<vec3<float> *>(&id_batch[block_size]);         // [block_size]
    vec3<S> *conic_batch =
        reinterpret_cast<vec3<float> *>(&xy_opacity_batch[block_size]); // [block_size]
    S *rgbs_batch = (S *)&conic_batch[block_size];                      // [block_size * COLOR_DIM]

    // this is the T AFTER the last gaussian in this pixel
    S T_final = 1.0f - render_alphas[pix_id];
    S T       = T_final;
    // the contribution from gaussians behind the current one
    S buffer[COLOR_DIM] = { 0.f };
    // index of last gaussian to contribute to this pixel
    const int32_t bin_final = inside ? last_ids[pix_id] : 0;

    // df/d_out for this pixel
    S v_render_c[COLOR_DIM];
    GSPLAT_PRAGMA_UNROLL
    for (uint32_t k = 0; k < COLOR_DIM; ++k) {
        v_render_c[k] = v_render_colors[pix_id * COLOR_DIM + k];
    }
    const S v_render_a = v_render_alphas[pix_id];

    // collect and process batches of gaussians
    // each thread loads one gaussian at a time before rasterizing
    const uint32_t            tr             = block.thread_rank();
    cg::thread_block_tile<32> warp           = cg::tiled_partition<32>(block);
    const int32_t             warp_bin_final = cg::reduce(warp, bin_final, cg::greater<int>());
    for (uint32_t b = 0; b < num_batches; ++b) {
        // resync all threads before writing next batch of shared mem
        block.sync();

        // each thread fetch 1 gaussian from back to front
        // 0 index will be furthest back in batch
        // index of gaussian to load
        // batch end is the index of the last gaussian in the batch
        // These values can be negative so must be int32 instead of uint32
        const int32_t batch_end  = range_end - 1 - block_size * b;
        const int32_t batch_size = min(block_size, batch_end + 1 - range_start);
        const int32_t idx        = batch_end - tr;
        if (idx >= range_start) {
            int32_t g            = flatten_ids[idx]; // flatten index in [C * N] or [nnz]
            id_batch[tr]         = g;
            const vec2<S> xy     = means2d[g];
            const S       opac   = opacities[g];
            xy_opacity_batch[tr] = { xy.x, xy.y, opac };
            conic_batch[tr]      = conics[g];
            if constexpr (N_OUTER_DIMS == 2) {
                // colors: [C, N, COLOR_DIM]
                // colors[c, n, k] = [c * N * COLOR_DIM + n * COLOR_DIM + k]
                // g = c * N + n
                const int32_t cid   = g / N;
                const int32_t gid   = g % N;
                const S      *c_ptr = colors[cid][gid].data();
                GSPLAT_PRAGMA_UNROLL
                for (uint32_t k = 0; k < COLOR_DIM; ++k) {
                    rgbs_batch[tr * COLOR_DIM + k] = c_ptr[k];
                }
            } else {
                const S *c_ptr = colors[g].data(); // + g * COLOR_DIM;
                GSPLAT_PRAGMA_UNROLL
                for (uint32_t k = 0; k < COLOR_DIM; ++k) {
                    rgbs_batch[tr * COLOR_DIM + k] = c_ptr[k];
                }
            }
        }
        // wait for other threads to collect the gaussians in batch
        block.sync();
        // process gaussians in the current batch for this pixel
        // 0 index is the furthest back gaussian in the batch
        for (uint32_t t = max(0, batch_end - warp_bin_final); t < batch_size; ++t) {
            bool valid = inside;
            if (batch_end - t > bin_final) {
                valid = 0;
            }
            S       alpha;
            S       opac;
            vec2<S> delta;
            vec3<S> conic;
            S       vis;

            if (valid) {
                conic           = conic_batch[t];
                vec3<S> xy_opac = xy_opacity_batch[t];
                opac            = xy_opac.z;
                delta           = { xy_opac.x - px, xy_opac.y - py };
                S sigma = 0.5f * (conic.x * delta.x * delta.x + conic.z * delta.y * delta.y) +
                          conic.y * delta.x * delta.y;
                vis   = __expf(-sigma);
                alpha = min(0.999f, opac * vis);
                if (sigma < 0.f || alpha < 1.f / 255.f) {
                    valid = false;
                }
            }

            // if all threads are inactive in this warp, skip this loop
            if (!warp.any(valid)) {
                continue;
            }
            S       v_rgb_local[COLOR_DIM] = { 0.f };
            vec3<S> v_conic_local          = { 0.f, 0.f, 0.f };
            vec2<S> v_xy_local             = { 0.f, 0.f };
            vec2<S> v_xy_abs_local         = { 0.f, 0.f };
            S       v_opacity_local        = 0.f;
            // initialize everything to 0, only set if the lane is valid
            if (valid) {
                // compute the current T for this gaussian
                S ra = 1.0f / (1.0f - alpha);
                T *= ra;
                // update v_rgb for this gaussian
                const S fac = alpha * T;
                GSPLAT_PRAGMA_UNROLL
                for (uint32_t k = 0; k < COLOR_DIM; ++k) {
                    v_rgb_local[k] = fac * v_render_c[k];
                }
                // contribution from this pixel
                S v_alpha = 0.f;
                for (uint32_t k = 0; k < COLOR_DIM; ++k) {
                    v_alpha += (rgbs_batch[t * COLOR_DIM + k] * T - buffer[k] * ra) * v_render_c[k];
                }

                v_alpha += T_final * ra * v_render_a;
                // contribution from background pixel
                if (backgrounds != nullptr) {
                    S accum = 0.f;
                    GSPLAT_PRAGMA_UNROLL
                    for (uint32_t k = 0; k < COLOR_DIM; ++k) {
                        accum += backgrounds[k] * v_render_c[k];
                    }
                    v_alpha += -T_final * ra * accum;
                }

                if (opac * vis <= 0.999f) {
                    const S v_sigma = -opac * vis * v_alpha;
                    v_conic_local   = { 0.5f * v_sigma * delta.x * delta.x,
                                        v_sigma * delta.x * delta.y,
                                        0.5f * v_sigma * delta.y * delta.y };
                    v_xy_local      = { v_sigma * (conic.x * delta.x + conic.y * delta.y),
                                        v_sigma * (conic.y * delta.x + conic.z * delta.y) };
                    if (v_means2d_abs != nullptr) {
                        v_xy_abs_local = { abs(v_xy_local.x), abs(v_xy_local.y) };
                    }
                    v_opacity_local = vis * v_alpha;
                }

                GSPLAT_PRAGMA_UNROLL
                for (uint32_t k = 0; k < COLOR_DIM; ++k) {
                    buffer[k] += rgbs_batch[t * COLOR_DIM + k] * fac;
                }
            }
            warpSum<COLOR_DIM, S>(v_rgb_local, warp);
            warpSum<decltype(warp), S>(v_conic_local, warp);
            warpSum<decltype(warp), S>(v_xy_local, warp);
            if (v_means2d_abs != nullptr) {
                warpSum<decltype(warp), S>(v_xy_abs_local, warp);
            }
            warpSum<decltype(warp), S>(v_opacity_local, warp);
            if (warp.thread_rank() == 0) {
                int32_t g         = id_batch[t]; // flatten index in [C * N] or [nnz]
                S      *v_rgb_ptr = (S *)(v_colors) + COLOR_DIM * g;
                GSPLAT_PRAGMA_UNROLL
                for (uint32_t k = 0; k < COLOR_DIM; ++k) {
                    gpuAtomicAdd(v_rgb_ptr + k, v_rgb_local[k]);
                }

                S *v_conic_ptr = (S *)(v_conics) + 3 * g;
                gpuAtomicAdd(v_conic_ptr, v_conic_local.x);
                gpuAtomicAdd(v_conic_ptr + 1, v_conic_local.y);
                gpuAtomicAdd(v_conic_ptr + 2, v_conic_local.z);

                S *v_xy_ptr = (S *)(v_means2d) + 2 * g;
                gpuAtomicAdd(v_xy_ptr, v_xy_local.x);
                gpuAtomicAdd(v_xy_ptr + 1, v_xy_local.y);

                if (v_means2d_abs != nullptr) {
                    S *v_xy_abs_ptr = (S *)(v_means2d_abs) + 2 * g;
                    gpuAtomicAdd(v_xy_abs_ptr, v_xy_abs_local.x);
                    gpuAtomicAdd(v_xy_abs_ptr + 1, v_xy_abs_local.y);
                }

                gpuAtomicAdd(v_opacities + g, v_opacity_local);
            }
        }
    }
}

template <uint32_t CDIM>
std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
call_bwd_kernel_with_dim(
    // Gaussian parameters
    const torch::Tensor               &means2d,     // [C, N, 2] or [nnz, 2]
    const torch::Tensor               &conics,      // [C, N, 3] or [nnz, 3]
    const torch::Tensor               &colors,      // [C, N, 3] or [nnz, 3]
    const torch::Tensor               &opacities,   // [C, N] or [nnz]
    const at::optional<torch::Tensor> &backgrounds, // [C, 3]
    const at::optional<torch::Tensor> &masks,       // [C, tile_height, tile_width]
    // image size
    const uint32_t image_width, const uint32_t image_height, const uint32_t image_origin_w,
    const uint32_t image_origin_h, const uint32_t tile_size,
    // intersections
    const torch::Tensor &tile_offsets, // [C, tile_height, tile_width]
    const torch::Tensor &flatten_ids,  // [n_isects]
    // forward outputs
    const torch::Tensor &render_alphas, // [C, image_height, image_width, 1]
    const torch::Tensor &last_ids,      // [C, image_height, image_width]
    // gradients of outputs
    const torch::Tensor &v_render_colors, // [C, image_height, image_width, 3]
    const torch::Tensor &v_render_alphas, // [C, image_height, image_width, 1]
    // options
    bool absgrad) {
    GSPLAT_DEVICE_GUARD(means2d);
    GSPLAT_CHECK_INPUT(means2d);
    GSPLAT_CHECK_INPUT(conics);
    GSPLAT_CHECK_CUDA(colors);
    GSPLAT_CHECK_INPUT(opacities);
    GSPLAT_CHECK_INPUT(tile_offsets);
    GSPLAT_CHECK_INPUT(flatten_ids);
    GSPLAT_CHECK_INPUT(render_alphas);
    GSPLAT_CHECK_INPUT(last_ids);
    GSPLAT_CHECK_INPUT(v_render_colors);
    GSPLAT_CHECK_INPUT(v_render_alphas);
    if (backgrounds.has_value()) {
        GSPLAT_CHECK_INPUT(backgrounds.value());
    }
    if (masks.has_value()) {
        GSPLAT_CHECK_INPUT(masks.value());
    }

    bool packed = means2d.dim() == 2;

    uint32_t C           = tile_offsets.size(0);         // number of cameras
    uint32_t N           = packed ? 0 : means2d.size(1); // number of gaussians
    uint32_t n_isects    = flatten_ids.size(0);
    uint32_t COLOR_DIM   = colors.size(-1);
    uint32_t tile_height = tile_offsets.size(1);
    uint32_t tile_width  = tile_offsets.size(2);

    const uint32_t tile_origin_w = image_origin_w / tile_size;
    const uint32_t tile_origin_h = image_origin_h / tile_size;
    const uint32_t tile_extent_w = (image_width + tile_size - 1) / tile_size;
    const uint32_t tile_extent_h = (image_height + tile_size - 1) / tile_size;

    // std::cerr << "RASTERIZE TO PIXELS BACKWARD " << std::endl;
    // std::cerr << "  BLOCKS = (" << C << ", " << tile_extent_h << ", " << tile_extent_w << ")"
    //           << std::endl;
    // std::cerr << "  THREADS = (" << tile_size << ", " << tile_size << ". " << 1 << ")" <<
    // std::endl; std::cerr << "  TILE WIDTH = " << tile_width << ", TILE HEIGHT = " << tile_height
    // << std::endl;

    // Each block covers a tile on the image. In total there are
    // C * tile_height * tile_width blocks.
    dim3 threads = { tile_size, tile_size, 1 };
    dim3 blocks  = { C, tile_extent_h, tile_extent_w };

    torch::Tensor v_means2d   = torch::zeros_like(means2d);
    torch::Tensor v_conics    = torch::zeros_like(conics);
    torch::Tensor v_colors    = torch::zeros_like(colors);
    torch::Tensor v_opacities = torch::zeros_like(opacities);
    torch::Tensor v_means2d_abs;
    if (absgrad) {
        v_means2d_abs = torch::zeros_like(means2d);
    }

    if (n_isects) {
        const uint32_t shared_mem = tile_size * tile_size *
                                    (sizeof(int32_t) + sizeof(vec3<float>) + sizeof(vec3<float>) +
                                     sizeof(float) * COLOR_DIM);
        at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();

        if (packed) {
            if (hipFuncSetAttribute(reinterpret_cast<const void*>(rasterize_to_pixels_bwd_kernel<CDIM), 1, float>,
                                     hipFuncAttributeMaxDynamicSharedMemorySize,
                                     shared_mem) != hipSuccess) {
                AT_ERROR("Failed to set maximum shared memory size (requested ", shared_mem,
                         " bytes), try lowering tile_size.");
            }
            rasterize_to_pixels_bwd_kernel<CDIM, 1, float><<<blocks, threads, shared_mem, stream>>>(
                C, N, n_isects, packed, reinterpret_cast<vec2<float> *>(means2d.data_ptr<float>()),
                reinterpret_cast<vec3<float> *>(conics.data_ptr<float>()),
                colors.packed_accessor64<float, 2, torch::RestrictPtrTraits>(),
                opacities.data_ptr<float>(),
                backgrounds.has_value() ? backgrounds.value().data_ptr<float>() : nullptr,
                masks.has_value() ? masks.value().data_ptr<bool>() : nullptr, image_width,
                image_height, image_origin_w, image_origin_h, tile_origin_w, tile_origin_h,
                tile_size, tile_width, tile_height, tile_offsets.data_ptr<int32_t>(),
                flatten_ids.data_ptr<int32_t>(), render_alphas.data_ptr<float>(),
                last_ids.data_ptr<int32_t>(), v_render_colors.data_ptr<float>(),
                v_render_alphas.data_ptr<float>(),
                absgrad ? reinterpret_cast<vec2<float> *>(v_means2d_abs.data_ptr<float>())
                        : nullptr,
                reinterpret_cast<vec2<float> *>(v_means2d.data_ptr<float>()),
                reinterpret_cast<vec3<float> *>(v_conics.data_ptr<float>()),
                v_colors.data_ptr<float>(), v_opacities.data_ptr<float>());
            C10_CUDA_KERNEL_LAUNCH_CHECK();
        } else {
            // int maxshmemperblock = 0;
            // hipDeviceGetAttribute(&maxshmemperblock, hipDeviceAttributeSharedMemPerBlockOptin,
            // 0); std::cerr << "maximum shared mem per block is " << maxshmemperblock << std::endl;
            if (hipFuncSetAttribute(reinterpret_cast<const void*>(rasterize_to_pixels_bwd_kernel<CDIM), 2, float>,
                                     hipFuncAttributeMaxDynamicSharedMemorySize,
                                     shared_mem) != hipSuccess) {
                AT_ERROR("Failed to set maximum shared memory size (requested ", shared_mem,
                         " bytes), try lowering tile_size.");
            }
            rasterize_to_pixels_bwd_kernel<CDIM, 2, float><<<blocks, threads, shared_mem, stream>>>(
                C, N, n_isects, packed, reinterpret_cast<vec2<float> *>(means2d.data_ptr<float>()),
                reinterpret_cast<vec3<float> *>(conics.data_ptr<float>()),
                colors.packed_accessor64<float, 3, torch::RestrictPtrTraits>(),
                opacities.data_ptr<float>(),
                backgrounds.has_value() ? backgrounds.value().data_ptr<float>() : nullptr,
                masks.has_value() ? masks.value().data_ptr<bool>() : nullptr, image_width,
                image_height, image_origin_w, image_origin_h, tile_origin_w, tile_origin_h,
                tile_size, tile_width, tile_height, tile_offsets.data_ptr<int32_t>(),
                flatten_ids.data_ptr<int32_t>(), render_alphas.data_ptr<float>(),
                last_ids.data_ptr<int32_t>(), v_render_colors.data_ptr<float>(),
                v_render_alphas.data_ptr<float>(),
                absgrad ? reinterpret_cast<vec2<float> *>(v_means2d_abs.data_ptr<float>())
                        : nullptr,
                reinterpret_cast<vec2<float> *>(v_means2d.data_ptr<float>()),
                reinterpret_cast<vec3<float> *>(v_conics.data_ptr<float>()),
                v_colors.data_ptr<float>(), v_opacities.data_ptr<float>());
            C10_CUDA_KERNEL_LAUNCH_CHECK();
        }
    }

    return std::make_tuple(v_means2d_abs, v_means2d, v_conics, v_colors, v_opacities);
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
rasterize_to_pixels_bwd_tensor(
    // Gaussian parameters
    const torch::Tensor               &means2d,     // [C, N, 2] or [nnz, 2]
    const torch::Tensor               &conics,      // [C, N, 3] or [nnz, 3]
    const torch::Tensor               &colors,      // [C, N, 3] or [nnz, 3]
    const torch::Tensor               &opacities,   // [C, N] or [nnz]
    const at::optional<torch::Tensor> &backgrounds, // [C, 3]
    const at::optional<torch::Tensor> &masks,       // [C, tile_height, tile_width]
    // image size
    const uint32_t image_width, const uint32_t image_height, const uint32_t image_origin_w,
    const uint32_t image_origin_h, const uint32_t tile_size,
    // intersections
    const torch::Tensor &tile_offsets, // [C, tile_height, tile_width]
    const torch::Tensor &flatten_ids,  // [n_isects]
    // forward outputs
    const torch::Tensor &render_alphas, // [C, image_height, image_width, 1]
    const torch::Tensor &last_ids,      // [C, image_height, image_width]
    // gradients of outputs
    const torch::Tensor &v_render_colors, // [C, image_height, image_width, 3]
    const torch::Tensor &v_render_alphas, // [C, image_height, image_width, 1]
    // options
    bool absgrad) {
    GSPLAT_CHECK_CUDA(colors);
    uint32_t COLOR_DIM = colors.size(-1);

#define __GS__CALL_BWD_(N)                                                                       \
    case N:                                                                                      \
        return call_bwd_kernel_with_dim<N>(                                                      \
            means2d, conics, colors, opacities, backgrounds, masks, image_width, image_height,   \
            image_origin_w, image_origin_h, tile_size, tile_offsets, flatten_ids, render_alphas, \
            last_ids, v_render_colors, v_render_alphas, absgrad);

    switch (COLOR_DIM) {
        __GS__CALL_BWD_(1)
        __GS__CALL_BWD_(2)
        __GS__CALL_BWD_(3)
        __GS__CALL_BWD_(4)
        __GS__CALL_BWD_(5)
        __GS__CALL_BWD_(8)
        __GS__CALL_BWD_(9)
        __GS__CALL_BWD_(16)
        __GS__CALL_BWD_(17)
        __GS__CALL_BWD_(32)
        __GS__CALL_BWD_(33)
        __GS__CALL_BWD_(64)
        __GS__CALL_BWD_(65)
        __GS__CALL_BWD_(128)
        __GS__CALL_BWD_(129)
        __GS__CALL_BWD_(256)
        __GS__CALL_BWD_(257)
        __GS__CALL_BWD_(512)
        __GS__CALL_BWD_(513)
    default:
        AT_ERROR("Unsupported number of channels: ", COLOR_DIM);
    }
}

template <>
std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
dispatchGaussianRasterizeBackward<torch::kCUDA>(
    // Gaussian parameters
    const torch::Tensor &means2d,   // [C, N, 2]
    const torch::Tensor &conics,    // [C, N, 3]
    const torch::Tensor &colors,    // [C, N, 3]
    const torch::Tensor &opacities, // [N]
    // image size
    const uint32_t image_width, const uint32_t image_height, const uint32_t image_origin_w,
    const uint32_t image_origin_h,

    const uint32_t tile_size,
    // intersections
    const torch::Tensor &tile_offsets, // [C, tile_height, tile_width]
    const torch::Tensor &flatten_ids,  // [n_isects]
    // forward outputs
    const torch::Tensor &render_alphas, // [C, image_height, image_width, 1]
    const torch::Tensor &last_ids,      // [C, image_height, image_width]
    // gradients of outputs
    const torch::Tensor &v_render_colors, // [C, image_height, image_width, 3]
    const torch::Tensor &v_render_alphas, // [C, image_height, image_width, 1]
    // options
    bool absgrad) {
    return rasterize_to_pixels_bwd_tensor(
        means2d, conics, colors, opacities, std::nullopt /*backgrounds*/, std::nullopt /*mask*/,
        image_width, image_height, image_origin_w, image_origin_h, tile_size, tile_offsets,
        flatten_ids, render_alphas, last_ids, v_render_colors, v_render_alphas, absgrad);
}

template <>
std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
dispatchGaussianRasterizeBackward<torch::kCPU>(
    // Gaussian parameters
    const torch::Tensor &means2d,   // [C, N, 2]
    const torch::Tensor &conics,    // [C, N, 3]
    const torch::Tensor &colors,    // [C, N, 3]
    const torch::Tensor &opacities, // [N]

    // image size
    const uint32_t image_width, const uint32_t image_height, const uint32_t image_origin_w,
    const uint32_t image_origin_h, const uint32_t tile_size,
    // intersections
    const torch::Tensor &tile_offsets, // [C, tile_height, tile_width]
    const torch::Tensor &flatten_ids,  // [n_isects]
    // forward outputs
    const torch::Tensor &render_alphas, // [C, image_height, image_width, 1]
    const torch::Tensor &last_ids,      // [C, image_height, image_width]
    // gradients of outputs
    const torch::Tensor &v_render_colors, // [C, image_height, image_width, 3]
    const torch::Tensor &v_render_alphas, // [C, image_height, image_width, 1]
    // options
    bool absgrad) {
    TORCH_CHECK_NOT_IMPLEMENTED(false, "CPU implementation not available");
}

} // namespace ops
} // namespace detail
} // namespace fvdb
