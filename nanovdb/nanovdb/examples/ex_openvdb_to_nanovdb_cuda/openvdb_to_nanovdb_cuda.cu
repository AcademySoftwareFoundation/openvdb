#include "hip/hip_runtime.h"
// Copyright Contributors to the OpenVDB Project
// SPDX-License-Identifier: MPL-2.0

#include <nanovdb/NanoVDB.h> // this defined the core tree data structure of NanoVDB accessable on both the host and device
#include <stdio.h> // for printf

// This is called by the host only
void cpu_kernel(const nanovdb::NanoGrid<float>* cpuGrid)
{
    printf("NanoVDB cpu; %4.2f\n", cpuGrid->tree().getValue(nanovdb::Coord(99, 0, 0)));
}

// This is called by the device only
__global__ void gpu_kernel(const nanovdb::NanoGrid<float>* deviceGrid)
{
    printf("NanoVDB gpu: %4.2f\n", deviceGrid->tree().getValue(nanovdb::Coord(99, 0, 0)));
}

// This is called by the client code on the host
extern "C" void launch_kernels(const nanovdb::NanoGrid<float>* deviceGrid,
                               const nanovdb::NanoGrid<float>* cpuGrid,
                               hipStream_t                    stream)
{
    gpu_kernel<<<1, 1, 0, stream>>>(deviceGrid); // Launch the device kernel asynchronously

    cpu_kernel(cpuGrid); // Launch the host "kernel" (synchronously)
}