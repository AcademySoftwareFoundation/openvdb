#include "hip/hip_runtime.h"
// Copyright Contributors to the OpenVDB Project
// SPDX-License-Identifier: MPL-2.0

#include <nanovdb/NanoVDB.h> // this defined the core tree data structure of NanoVDB accessable on both the host and device
#include <nanovdb/util/cuda/CudaGridHandle.cuh>// required since GridHandle<CudaDeviceBuffer> has device code
#include <stdio.h> // for printf

// This is called by the host only
void cpu_kernel(const nanovdb::NanoGrid<float>* cpuGrid)
{
    auto cpuAcc = cpuGrid->getAccessor();
    for (int k=-3; k<=3; k+=6) {
        printf("NanoVDB cpu: (%i,%i,%i)=%4.2f\n", 1, 2, k, cpuAcc.getValue(nanovdb::Coord(1, 2, k)));
    }
}

// This is called by the device only
__global__ void gpu_kernel(const nanovdb::NanoGrid<float>* deviceGrid)
{
    if (threadIdx.x != 0 && threadIdx.x != 6) return;
    int k = threadIdx.x - 3;
    auto gpuAcc = deviceGrid->getAccessor();
    printf("NanoVDB gpu: (%i,%i,%i)=%4.2f\n", 1, 2, k, gpuAcc.getValue(nanovdb::Coord(1, 2, k)));
}

// This is called by the client code on the host
extern "C" void launch_kernels(const nanovdb::NanoGrid<float>* deviceGrid,
                               const nanovdb::NanoGrid<float>* cpuGrid,
                               hipStream_t                    stream)
{
    // Launch the device kernel asynchronously
    gpu_kernel<<<1, 64, 0, stream>>>(deviceGrid);

    // Launch the host "kernel" (synchronously)
    cpu_kernel(cpuGrid);
}
