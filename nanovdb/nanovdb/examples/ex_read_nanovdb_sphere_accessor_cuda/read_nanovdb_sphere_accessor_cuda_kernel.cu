#include "hip/hip_runtime.h"
// Copyright Contributors to the OpenVDB Project
// SPDX-License-Identifier: MPL-2.0

#include <nanovdb/NanoVDB.h> // this defined the core tree data structure of NanoVDB accessable on both the host and device
#include <stdio.h> // for printf

// This is called by the host only
void cpu_kernel(const nanovdb::NanoGrid<float>* cpuGrid)
{
    auto cpuAcc = cpuGrid->getAccessor();
    for (int i = 97; i < 104; ++i) {
        printf("(%3i,0,0) NanoVDB cpu: % -4.2f\n", i, cpuAcc.getValue(nanovdb::Coord(i, 0, 0)));
    }
}

// This is called by the device only
__global__ void gpu_kernel(const nanovdb::NanoGrid<float>* deviceGrid)
{
    if (threadIdx.x > 6)
        return;
    int  i = 97 + threadIdx.x;
    auto gpuAcc = deviceGrid->getAccessor();
    printf("(%3i,0,0) NanoVDB gpu: % -4.2f\n", i, gpuAcc.getValue(nanovdb::Coord(i, 0, 0)));
}

// This is called by the client code on the host
extern "C" void launch_kernels(const nanovdb::NanoGrid<float>* deviceGrid,
                               const nanovdb::NanoGrid<float>* cpuGrid,
                               hipStream_t                    stream)
{
    // Launch the device kernel asynchronously
    gpu_kernel<<<1, 64, 0, stream>>>(deviceGrid);

    // Launch the host "kernel" (synchronously)
    cpu_kernel(cpuGrid);
}