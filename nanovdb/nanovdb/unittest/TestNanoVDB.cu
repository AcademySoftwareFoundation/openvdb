#include "hip/hip_runtime.h"
// Copyright Contributors to the OpenVDB Project
// SPDX-License-Identifier: MPL-2.0

#include <vector>
#include <nanovdb/NanoVDB.h>
#include <nanovdb/util/ForEach.h>
#include <nanovdb/util/GridBuilder.h>
#include <nanovdb/util/CreateNanoGrid.h>
#include <nanovdb/util/Primitives.h>
#include <nanovdb/util/cuda/CudaUtils.h>
#include <nanovdb/util/cuda/CudaSignedFloodFill.h>
#include <nanovdb/util/cuda/CudaPointsToGrid.h>
#include <nanovdb/util/cuda/CudaIndexToGrid.h>
#include <nanovdb/util/cuda/CudaAddBlindData.h>
#include <nanovdb/util/CpuTimer.h>
#include <nanovdb/util/IO.h>

#include <gtest/gtest.h>

namespace nanovdb {// this namespace is required by gtest

namespace test {
// used for testing CudaDeviceBuffer
void device2host(size_t count)
{
    const size_t size = count * sizeof(float);
    auto buffer = nanovdb::CudaDeviceBuffer::create(size, nullptr, false);// on device only
    EXPECT_EQ(size, buffer.size());
    EXPECT_FALSE(buffer.data());
    EXPECT_TRUE(buffer.deviceData());
    float *d_array = reinterpret_cast<float*>(buffer.deviceData());
    constexpr unsigned int num_threads = 256;
    unsigned int num_blocks = num_blocks = (static_cast<unsigned int>(count) + num_threads - 1) / num_threads;
    cudaLambdaKernel<<<num_blocks, num_threads>>>(count, [=] __device__ (size_t i) {d_array[i] = float(i);});
    buffer.deviceDownload();// copy device -> host
    EXPECT_EQ(size, buffer.size());
    EXPECT_TRUE(buffer.data());
    EXPECT_TRUE(buffer.deviceData());
    float *array = reinterpret_cast<float*>(buffer.data());
    for (size_t i=0; i<count; ++i) EXPECT_EQ(array[i], float(i));
}// device2host
// used for testing CudaDeviceBuffer
void host2device2host(size_t count)
{
    bool *test, *d_test;
    cudaCheck(hipHostMalloc((void**)&test, sizeof(bool)));
    cudaCheck(hipMalloc((void**)&d_test, sizeof(bool)));
    *test = true;
    cudaCheck(hipMemcpyAsync(d_test, test, sizeof(bool), hipMemcpyHostToDevice));// on host only

    const size_t size = count * sizeof(float);
    auto buffer = nanovdb::CudaDeviceBuffer::create(size);
    EXPECT_EQ(size, buffer.size());
    EXPECT_TRUE(buffer.data());
    EXPECT_FALSE(buffer.deviceData());
    float *array = reinterpret_cast<float*>(buffer.data());
    for (size_t i=0; i<count; ++i) array[i] = i;
    buffer.deviceUpload();// copy host -> device
    EXPECT_EQ(size, buffer.size());
    EXPECT_TRUE(buffer.data());
    EXPECT_TRUE(buffer.deviceData());
    float *d_array = reinterpret_cast<float*>(buffer.deviceData());
    constexpr unsigned int num_threads = 256;
    unsigned int num_blocks = num_blocks = (static_cast<unsigned int>(count) + num_threads - 1) / num_threads;
    cudaLambdaKernel<<<num_blocks, num_threads>>>(count, [=] __device__ (size_t i) {
        if (d_array[i] != float(i)) *d_test = false;
        d_array[i] = float(i) + 1.0f;
    });
    cudaCheck(hipMemcpy(test, d_test, sizeof(bool), hipMemcpyDeviceToHost));
    EXPECT_TRUE(*test);
    cudaCheck(hipHostFree(test));
    cudaCheck(hipFree(d_test));
    buffer.deviceDownload();// copy device -> host
    EXPECT_EQ(size, buffer.size());
    EXPECT_TRUE(buffer.data());
    EXPECT_TRUE(buffer.deviceData());
    for (size_t i=0; i<count; ++i) EXPECT_EQ(array[i], float(i) + 1.0f);
}// host2device2host
// used to test cudaStr methods
void cudaStr()
{
    const size_t size = 50;
    char *str, *d_str;
    cudaCheck(hipHostMalloc((void**)&str, size));
    cudaCheck(hipMalloc(  (void**)&d_str, size));
    int n, *d_n;
    cudaCheck(hipMalloc((void**)&d_n, sizeof(int)));

    cudaLambdaKernel<<<1, 1>>>(1, [=] __device__ (size_t) {
        cudaStrcpy(d_str, "this is a test");
    });
    cudaCheck(hipMemcpy(str, d_str, size, hipMemcpyDeviceToHost));
    EXPECT_STREQ(str, "this is a test");
    cudaLambdaKernel<<<1, 1>>>(1, [=] __device__ (size_t) {
        cudaStrcat(d_str, " #2");
    });
    cudaCheck(hipMemcpy(str, d_str, size, hipMemcpyDeviceToHost));
    EXPECT_STREQ(str, "this is a test #2");

    cudaLambdaKernel<<<1, 1>>>(1, [=] __device__ (size_t) {
        *d_n = cudaStrcmp(d_str, "this is a test");
    });
    cudaCheck(hipMemcpy(&n, d_n, sizeof(int), hipMemcpyDeviceToHost));
    //std::cerr << "n = " << n << std::endl;
    EXPECT_EQ(std::strcmp(str, "this is a test"), n);
    cudaLambdaKernel<<<1, 1>>>(1, [=] __device__ (size_t) {
        *d_n = cudaStrcmp(d_str, "this is a test #2");
    });
    cudaCheck(hipMemcpy(&n, d_n, sizeof(int), hipMemcpyDeviceToHost));
    EXPECT_EQ(std::strcmp(str, "this is a test #2"), n);
    EXPECT_EQ(0, n);

    cudaCheck(hipHostFree(str));
    cudaCheck(hipFree(d_n));
    cudaCheck(hipFree(d_str));
}// cudaStr
}// namespace test
}// namespace nanovdb

TEST(TestNanoVDBCUDA, CudaDeviceBuffer)
{
   nanovdb::test::device2host(1000);
   nanovdb::test::host2device2host(1000);
}

TEST(TestNanoVDBCUDA, CudaStr)
{
   nanovdb::test::cudaStr();
}

TEST(TestNanoVDBCUDA, Basic_CudaPointsToGrid_float)
{
    using BuildT = float;
    using GridT = nanovdb::NanoGrid<BuildT>;
    const size_t num_points = 1;
    nanovdb::Coord coords[num_points] = {nanovdb::Coord(1, 2, 3)}, *d_coords = nullptr;
    cudaCheck(hipMalloc(&d_coords, num_points * sizeof(nanovdb::Coord)));
    cudaCheck(hipMemcpy(d_coords, coords, num_points * sizeof(nanovdb::Coord), hipMemcpyHostToDevice));// CPU -> GPU

    nanovdb::CudaPointsToGrid<BuildT> converter;
    auto handle = converter.getHandle(d_coords, num_points);
    cudaCheck(hipFree(d_coords));
    EXPECT_TRUE(handle.deviceData());// grid only exists on the GPU
    EXPECT_FALSE(handle.data());// no grid was yet allocated on the CPU

    const uint64_t size = sizeof(GridT) +
                          sizeof(GridT::TreeType) +
                          GridT::RootType::memUsage(1) +
                          sizeof(GridT::UpperNodeType) +
                          sizeof(GridT::LowerNodeType) +
                          sizeof(GridT::LeafNodeType);
    EXPECT_EQ(handle.size(), size);

    GridT *grid = handle.grid<BuildT>();// no grid on the CPU
    EXPECT_FALSE(grid);
    handle.deviceDownload();// creates a copy up the CPU
    EXPECT_TRUE(handle.deviceData());
    EXPECT_TRUE(handle.data());
    auto *data = handle.gridData();
    EXPECT_TRUE(data);
    grid = handle.grid<BuildT>();
    EXPECT_TRUE(grid);

    auto acc = grid->getAccessor();
    EXPECT_FALSE(acc.isActive(nanovdb::Coord(0,2,3)));
    EXPECT_TRUE( acc.isActive(nanovdb::Coord(1,2,3)));
    EXPECT_FALSE(acc.isActive(nanovdb::Coord(1,2,4)));
    EXPECT_FALSE(acc.isActive(nanovdb::Coord(2,2,3)));
    auto *leaf = acc.probeLeaf(nanovdb::Coord(1,2,3));
    EXPECT_TRUE(leaf);
    EXPECT_EQ(nanovdb::Coord(0), leaf->origin());
    EXPECT_EQ(1u, leaf->valueMask().countOn());
    EXPECT_EQ(nanovdb::Coord(1,2,3), leaf->bbox()[0]);
    EXPECT_EQ(nanovdb::Coord(1,2,3), leaf->bbox()[1]);
}// Basic_CudaPointsToGrid_float

namespace nanovdb {
namespace test {

/// @brief Implements Tree::probeValue(Coord)
/// @tparam BuildT Build type of the grid being called
template <typename BuildT>
struct ProbeValueNew {
    using ValueT = typename BuildToValueMap<BuildT>::Type;
    struct Probe {
        bool state;
        ValueT value;
        operator bool() const { return state; }
    };
    __hostdev__ static Probe get(const NanoRoot<BuildT>  &root) {
        return Probe{false, root.mBackground};
    }
    __hostdev__ static Probe get(const typename NanoRoot<BuildT>::Tile &tile) {
        return Probe{tile.state>0, tile.value};
    }
    __hostdev__ static Probe get(const NanoUpper<BuildT> &node, uint32_t n) {
        return Probe{node.mValueMask.isOn(n), node.mTable[n].value};
    }
    __hostdev__ static Probe get(const NanoLower<BuildT> &node, uint32_t n) {
        return Probe{node.mValueMask.isOn(n), node.mTable[n].value};
    }
    __hostdev__ static Probe get(const NanoLeaf<BuildT>  &leaf, uint32_t n) {
        return Probe{leaf.isActive(n), leaf.getValue(n)};
    }
};// ProbeValueNew<BuildT>

template <typename BuildT>
struct AccessLeafMask;

// template specialization of AccessLeafMask wrt ValueOnIndexMask
template <>
struct AccessLeafMask<ValueOnIndexMask>{
    __hostdev__ static bool get(const NanoRoot<ValueOnIndexMask>&) {return false;}
    __hostdev__ static bool get(const typename NanoRoot<ValueOnIndexMask>::Tile&) {return false;}
    __hostdev__ static bool get(const NanoUpper<ValueOnIndexMask>&, uint32_t) {return false;}
    __hostdev__ static bool get(const NanoLower<ValueOnIndexMask>&, uint32_t) {return false;}
    __hostdev__ static bool get(const NanoLeaf<ValueOnIndexMask> &leaf, uint32_t n) {return leaf.mMask.isOn(n);}
    __hostdev__ static void set(NanoRoot<ValueOnIndexMask>&) {}
    __hostdev__ static void set(typename NanoRoot<ValueOnIndexMask>::Tile&) {}
    __hostdev__ static void set(NanoUpper<ValueOnIndexMask>&, uint32_t) {}
    __hostdev__ static void set(NanoLower<ValueOnIndexMask>&, uint32_t) {}
    __hostdev__ static void set(NanoLeaf<ValueOnIndexMask> &leaf, uint32_t n) {leaf.mMask.setOn(n);}
};// AccessLeafMask<BuildT>

}// end of test namespace
}// end of nanovdb namespace

TEST(TestNanoVDBCUDA, Basic_CudaPointsToGrid_ValueIndex)
{
    using BuildT = nanovdb::ValueIndex;
    using GridT = nanovdb::NanoGrid<BuildT>;
    const size_t num_points = 3;
    nanovdb::Coord coords[num_points] = {nanovdb::Coord(1, 2, 3),
                                         nanovdb::Coord(1, 2, 4),
                                         nanovdb::Coord(8, 2, 3)}, *d_coords = nullptr;
    cudaCheck(hipMalloc(&d_coords, num_points * sizeof(nanovdb::Coord)));
    cudaCheck(hipMemcpy(d_coords, coords, num_points * sizeof(nanovdb::Coord), hipMemcpyHostToDevice));// CPU -> GPU
#if 0
    nanovdb::CudaPointsToGrid converter;
    auto handle = converter.getHandle<BuildT>(d_coords, num_points);
#else
    auto handle = nanovdb::cudaVoxelsToGrid<BuildT>(d_coords, num_points);
#endif
    cudaCheck(hipFree(d_coords));
    EXPECT_TRUE(handle.deviceData());// grid only exists on the GPU
    EXPECT_FALSE(handle.data());// no grid was yet allocated on the CPU

    const uint64_t size = sizeof(GridT) +
                          sizeof(GridT::TreeType) +
                          GridT::RootType::memUsage(1) +
                          sizeof(GridT::UpperNodeType) +
                          sizeof(GridT::LowerNodeType) +
                          2*sizeof(GridT::LeafNodeType);
    EXPECT_EQ(handle.size(), size);

    GridT *grid = handle.grid<BuildT>();// no grid on the CPU
    EXPECT_FALSE(grid);
    handle.deviceDownload();// creates a copy up the CPU
    EXPECT_TRUE(handle.deviceData());
    EXPECT_TRUE(handle.data());
    auto *data = handle.gridData();
    EXPECT_TRUE(data);
    grid = handle.grid<BuildT>();
    EXPECT_TRUE(grid);
    EXPECT_EQ(1u + 2*512u, grid->valueCount());

    auto acc = grid->getAccessor();
    EXPECT_FALSE( acc.isActive(nanovdb::Coord(0,2,3)));
    EXPECT_TRUE(  acc.isActive(nanovdb::Coord(1,2,3)));
    EXPECT_TRUE(  acc.isActive(nanovdb::Coord(1,2,4)));
    EXPECT_TRUE(  acc.isActive(nanovdb::Coord(8,2,3)));
    EXPECT_EQ(1u + nanovdb::NanoLeaf<BuildT>::CoordToOffset(nanovdb::Coord(0,2,3)), acc.getValue(nanovdb::Coord(0,2,3)));
    EXPECT_EQ(1u + nanovdb::NanoLeaf<BuildT>::CoordToOffset(nanovdb::Coord(1,2,3)), acc.getValue(nanovdb::Coord(1,2,3)));
    EXPECT_EQ(1u + nanovdb::NanoLeaf<BuildT>::CoordToOffset(nanovdb::Coord(2,2,3)), acc.getValue(nanovdb::Coord(2,2,3)));
    EXPECT_EQ(1u + 512u + nanovdb::NanoLeaf<BuildT>::CoordToOffset(nanovdb::Coord(8,2,3)), acc.getValue(nanovdb::Coord(8,2,3)));

    using OpT = nanovdb::GetValue<BuildT>;
    EXPECT_EQ(1u + nanovdb::NanoLeaf<BuildT>::CoordToOffset(nanovdb::Coord(0,2,3)), acc.get<OpT>(nanovdb::Coord(0,2,3)));
    EXPECT_EQ(1u + nanovdb::NanoLeaf<BuildT>::CoordToOffset(nanovdb::Coord(1,2,3)), acc.get<OpT>(nanovdb::Coord(1,2,3)));
    EXPECT_EQ(1u + nanovdb::NanoLeaf<BuildT>::CoordToOffset(nanovdb::Coord(2,2,3)), acc.get<OpT>(nanovdb::Coord(2,2,3)));
    EXPECT_EQ(1u + 512u + nanovdb::NanoLeaf<BuildT>::CoordToOffset(nanovdb::Coord(8,2,3)), acc.get<OpT>(nanovdb::Coord(8,2,3)));

    for (size_t i=0; i<num_points; ++i)  {
        const nanovdb::Coord ijk = coords[i];
        const auto *leaf = acc.get<nanovdb::GetLeaf<BuildT>>(ijk);
        EXPECT_TRUE(leaf);
        const auto offset = leaf->CoordToOffset(ijk);
        EXPECT_EQ(ijk, leaf->offsetToGlobalCoord(offset));
    }
}// Basic_CudaPointsToGrid_ValueIndex

TEST(TestNanoVDBCUDA, Basic_CudaPointsToGrid_ValueOnIndex)
{
    using BuildT = nanovdb::ValueOnIndex;
    using GridT = nanovdb::NanoGrid<BuildT>;
    EXPECT_TRUE(nanovdb::BuildTraits<BuildT>::is_index);
    EXPECT_FALSE(nanovdb::BuildTraits<BuildT>::is_indexmask);
    EXPECT_TRUE(nanovdb::BuildTraits<BuildT>::is_onindex);
    EXPECT_FALSE(nanovdb::BuildTraits<BuildT>::is_offindex);
    const size_t num_points = 3;
    nanovdb::Coord coords[num_points] = {nanovdb::Coord(1, 2, 3),
                                         nanovdb::Coord(1, 2, 4),
                                         nanovdb::Coord(8, 2, 3)}, *d_coords = nullptr;
    cudaCheck(hipMalloc(&d_coords, num_points * sizeof(nanovdb::Coord)));
    cudaCheck(hipMemcpy(d_coords, coords, num_points * sizeof(nanovdb::Coord), hipMemcpyHostToDevice));// CPU -> GPU

#if 0
    nanovdb::CudaPointsToGrid converter;
    auto handle = converter.getHandle<BuildT>(d_coords, num_points);
#else
    auto handle = nanovdb::cudaVoxelsToGrid<BuildT>(d_coords, num_points);
#endif

    cudaCheck(hipFree(d_coords));
    EXPECT_TRUE(handle.deviceData());// grid only exists on the GPU
    EXPECT_FALSE(handle.data());// no grid was yet allocated on the CPU

    const uint64_t size = sizeof(GridT) +
                          sizeof(GridT::TreeType) +
                          GridT::RootType::memUsage(1) +
                          sizeof(GridT::UpperNodeType) +
                          sizeof(GridT::LowerNodeType) +
                          2*sizeof(GridT::LeafNodeType);
    EXPECT_EQ(handle.size(), size);

    GridT *grid = handle.grid<BuildT>();// no grid on the CPU
    EXPECT_FALSE(grid);
    handle.deviceDownload();// creates a copy up the CPU
    EXPECT_TRUE(handle.deviceData());
    EXPECT_TRUE(handle.data());
    auto *data = handle.gridData();
    EXPECT_TRUE(data);
    grid = handle.grid<BuildT>();
    EXPECT_TRUE(grid);
    EXPECT_EQ(1u + num_points, grid->valueCount());

    auto acc = grid->getAccessor();
    EXPECT_FALSE( acc.isActive(nanovdb::Coord(0,2,3)));
    EXPECT_TRUE(  acc.isActive(nanovdb::Coord(1,2,3)));
    EXPECT_TRUE(  acc.isActive(nanovdb::Coord(1,2,4)));
    EXPECT_TRUE(  acc.isActive(nanovdb::Coord(8,2,3)));
    EXPECT_EQ(0u, acc.getValue(nanovdb::Coord(0,2,3)));
    EXPECT_EQ(1u, acc.getValue(nanovdb::Coord(1,2,3)));
    EXPECT_EQ(2u, acc.getValue(nanovdb::Coord(1,2,4)));
    EXPECT_EQ(3u, acc.getValue(nanovdb::Coord(8,2,3)));

    using GetT = nanovdb::GetValue<BuildT>;
    EXPECT_EQ(0u, acc.get<GetT>(nanovdb::Coord(0,2,3)));
    EXPECT_EQ(1u, acc.get<GetT>(nanovdb::Coord(1,2,3)));
    EXPECT_EQ(2u, acc.get<GetT>(nanovdb::Coord(1,2,4)));
    EXPECT_EQ(3u, acc.get<GetT>(nanovdb::Coord(8,2,3)));

    {
        using T = nanovdb::test::ProbeValueNew<BuildT>;
        auto tmp = acc.get<T>(nanovdb::Coord(0,2,3));
        EXPECT_EQ(false, tmp.state);
        EXPECT_EQ(0u,    tmp.value);
        tmp = acc.get<T>(nanovdb::Coord(1,2,3));
        EXPECT_EQ(true, tmp.state);
        EXPECT_EQ(1u,   tmp.value);
        tmp = acc.get<T>(nanovdb::Coord(1,2,4));
        EXPECT_EQ(true, tmp.state);
        EXPECT_EQ(2u,   tmp.value);
        tmp = acc.get<T>(nanovdb::Coord(8,2,3));
        EXPECT_EQ(true, tmp.state);
        EXPECT_EQ(3u,   tmp.value);
    }
    {
        using T = nanovdb::ProbeValue<BuildT>;
        uint64_t value = 0;
        EXPECT_EQ(false, acc.get<T>(nanovdb::Coord(0,2,3), value) );
        EXPECT_EQ(0u,    value);
        EXPECT_EQ(true, acc.get<T>(nanovdb::Coord(1,2,3), value) );
        EXPECT_EQ(1u,    value);
        EXPECT_EQ(true, acc.get<T>(nanovdb::Coord(1,2,4), value) );
        EXPECT_EQ(2u,    value);
        EXPECT_EQ(true, acc.get<T>(nanovdb::Coord(8,2,3), value) );
        EXPECT_EQ(3u,    value);
    }

    for (size_t i=0; i<num_points; ++i)  {
        const nanovdb::Coord ijk = coords[i];
        const auto *leaf = acc.get<nanovdb::GetLeaf<BuildT>>(ijk);
        EXPECT_TRUE(leaf);
        const auto offset = leaf->CoordToOffset(ijk);
        EXPECT_EQ(ijk, leaf->offsetToGlobalCoord(offset));
    }
}// Basic_CudaPointsToGrid_ValueOnIndex

TEST(TestNanoVDBCUDA, Basic_CudaPointsToGrid_ValueOnIndexMask)
{
    using BuildT = nanovdb::ValueOnIndexMask;
    using GridT = nanovdb::NanoGrid<BuildT>;
    EXPECT_TRUE(nanovdb::BuildTraits<BuildT>::is_index);
    EXPECT_TRUE(nanovdb::BuildTraits<BuildT>::is_indexmask);
    EXPECT_TRUE(nanovdb::BuildTraits<BuildT>::is_onindex);
    EXPECT_FALSE(nanovdb::BuildTraits<BuildT>::is_offindex);
    const size_t num_points = 3;
    nanovdb::Coord coords[num_points] = {nanovdb::Coord(1, 2, 3),
                                         nanovdb::Coord(1, 2, 4),
                                         nanovdb::Coord(8, 2, 3)}, *d_coords = nullptr;
    cudaCheck(hipMalloc(&d_coords, num_points * sizeof(nanovdb::Coord)));
    cudaCheck(hipMemcpy(d_coords, coords, num_points * sizeof(nanovdb::Coord), hipMemcpyHostToDevice));// CPU -> GPU

#if 0
    nanovdb::CudaPointsToGrid converter;
    auto handle = converter.getHandle<BuildT>(d_coords, num_points);
#else
    auto handle = nanovdb::cudaVoxelsToGrid<BuildT>(d_coords, num_points);
#endif

    cudaCheck(hipFree(d_coords));
    EXPECT_TRUE(handle.deviceData());// grid only exists on the GPU
    EXPECT_FALSE(handle.data());// no grid was yet allocated on the CPU

    const uint64_t size = sizeof(GridT) +
                          sizeof(GridT::TreeType) +
                          GridT::RootType::memUsage(1) +
                          sizeof(GridT::UpperNodeType) +
                          sizeof(GridT::LowerNodeType) +
                          2*sizeof(GridT::LeafNodeType);
    EXPECT_EQ(handle.size(), size);

    GridT *grid = handle.grid<BuildT>();// no grid on the CPU
    EXPECT_FALSE(grid);
    handle.deviceDownload();// creates a copy up the CPU
    EXPECT_TRUE(handle.deviceData());
    EXPECT_TRUE(handle.data());
    auto *data = handle.gridData();
    EXPECT_TRUE(data);
    grid = handle.grid<BuildT>();
    EXPECT_TRUE(grid);
    EXPECT_EQ(4u, grid->valueCount());

    auto acc = grid->getAccessor();
    EXPECT_FALSE( acc.isActive(nanovdb::Coord(0,2,3)));
    EXPECT_TRUE(  acc.isActive(nanovdb::Coord(1,2,3)));
    EXPECT_TRUE(  acc.isActive(nanovdb::Coord(1,2,4)));
    EXPECT_TRUE(  acc.isActive(nanovdb::Coord(8,2,3)));
    EXPECT_EQ(0u, acc.getValue(nanovdb::Coord(0,2,3)));
    EXPECT_EQ(1u, acc.getValue(nanovdb::Coord(1,2,3)));
    EXPECT_EQ(2u, acc.getValue(nanovdb::Coord(1,2,4)));
    EXPECT_EQ(3u, acc.getValue(nanovdb::Coord(8,2,3)));

    using GetT = nanovdb::GetValue<BuildT>;
    EXPECT_EQ(0u, acc.get<GetT>(nanovdb::Coord(0,2,3)));
    EXPECT_EQ(1u, acc.get<GetT>(nanovdb::Coord(1,2,3)));
    EXPECT_EQ(2u, acc.get<GetT>(nanovdb::Coord(1,2,4)));
    EXPECT_EQ(3u, acc.get<GetT>(nanovdb::Coord(8,2,3)));

    using OpT = nanovdb::test::AccessLeafMask<BuildT>;
    EXPECT_EQ(false, acc.get<OpT>(nanovdb::Coord(0,2,3)));
    EXPECT_EQ(true,  acc.get<OpT>(nanovdb::Coord(1,2,3)));
    EXPECT_EQ(true,  acc.get<OpT>(nanovdb::Coord(1,2,4)));
    EXPECT_EQ(true,  acc.get<OpT>(nanovdb::Coord(8,2,3)));

    acc.set<OpT>(nanovdb::Coord(1,2,3));
    acc.set<OpT>(nanovdb::Coord(8,2,3));

    EXPECT_EQ(false, acc.get<OpT>(nanovdb::Coord(0,2,3)));
    EXPECT_EQ(true , acc.get<OpT>(nanovdb::Coord(1,2,3)));
    EXPECT_EQ(true,  acc.get<OpT>(nanovdb::Coord(1,2,4)));
    EXPECT_EQ(true,  acc.get<OpT>(nanovdb::Coord(8,2,3)));

    {
        using T = nanovdb::ProbeValue<BuildT>;
        uint64_t value = 0;
        EXPECT_EQ(false, acc.get<T>(nanovdb::Coord(0,2,3), value) );
        EXPECT_EQ(0u,    value);
        EXPECT_EQ(true,  acc.get<T>(nanovdb::Coord(1,2,3), value) );
        EXPECT_EQ(1u,    value);
        EXPECT_EQ(true,  acc.get<T>(nanovdb::Coord(1,2,4), value) );
        EXPECT_EQ(2u,    value);
        EXPECT_EQ(true,  acc.get<T>(nanovdb::Coord(8,2,3), value) );
        EXPECT_EQ(3u,    value);
        EXPECT_EQ(false, acc.get<T>(nanovdb::Coord(-18,2,3), value) );
        EXPECT_EQ(0u,    value);

        EXPECT_EQ(false, grid->tree().get<T>(nanovdb::Coord(0,2,3), value) );
        EXPECT_EQ(0u,    value);
        EXPECT_EQ(true,  grid->tree().get<T>(nanovdb::Coord(1,2,3), value) );
        EXPECT_EQ(1u,    value);
        EXPECT_EQ(true,  grid->tree().get<T>(nanovdb::Coord(1,2,4), value) );
        EXPECT_EQ(2u,    value);
        EXPECT_EQ(true,  grid->tree().get<T>(nanovdb::Coord(8,2,3), value) );
        EXPECT_EQ(3u,    value);
        EXPECT_EQ(false, grid->tree().get<T>(nanovdb::Coord(-18,2,3), value) );
        EXPECT_EQ(0u,    value);
    }

    for (size_t i=0; i<num_points; ++i)  {
        const nanovdb::Coord ijk = coords[i];
        const auto *leaf = acc.get<nanovdb::GetLeaf<BuildT>>(ijk);
        EXPECT_TRUE(leaf);
        const auto offset = leaf->CoordToOffset(ijk);
        EXPECT_EQ(ijk, leaf->offsetToGlobalCoord(offset));
        EXPECT_EQ(leaf->mValueMask, leaf->mMask);
    }
}// Basic_CudaPointsToGrid_ValueOnIndexMask

TEST(TestNanoVDBCUDA, Large_CudaPointsToGrid_old)
{
    using BuildT = nanovdb::ValueOnIndex;
    nanovdb::CpuTimer timer;
    const size_t voxelCount = 1 << 20;// 1048576
    std::vector<nanovdb::Coord> voxels;
    {//generate random voxels
        voxels.reserve(voxelCount);
        std::srand(98765);
        const int max = 512, min = -max;
        auto op = [&](){return rand() % (max - min) + min;};
        timer.start("Creating "+std::to_string(voxelCount)+" random voxels on the CPU");
        while (voxels.size() < voxelCount) voxels.push_back(nanovdb::Coord(op(), op(), op()));
        timer.stop();
        EXPECT_EQ(voxelCount, voxels.size());
    }
#if 0
    {// Build grid on CPU
        nanovdb::build::Grid<float> buildGrid(0.0f);
        timer.start("Building grid on CPU from "+std::to_string(voxels.size())+" points");
        nanovdb::forEach(0, voxelCount, voxelCount >> 6, [&](const nanovdb::Range1D &r){
            auto acc = buildGrid.getWriteAccessor();
            for (size_t i=r.begin(); i!=r.end(); ++i) acc.setValueOn(voxels[i]);
        });
        timer.restart("Converting CPU build::Grid to nanovdb");
        auto handle = nanovdb::createNanoGrid(buildGrid);
        timer.stop();
    }
#endif
    nanovdb::Coord* d_coords;
    const size_t voxelSize = voxels.size() * sizeof(nanovdb::Coord);
    //timer.start("Allocating "+std::to_string(voxelSize >> 20)+" MB on the GPU");
    cudaCheck(hipMalloc(&d_coords, voxelSize));
    //timer.restart("Copying voxels from CPU to GPU");
    cudaCheck(hipMemcpy(d_coords, voxels.data(), voxelSize, hipMemcpyHostToDevice));
    //timer.stop();

    timer.start("Building grid on GPU from "+std::to_string(voxels.size())+" points");
    nanovdb::CudaPointsToGrid<BuildT> converter;
    //converter.setVerbose();
    auto handle = converter.getHandle(d_coords, voxelCount);
    timer.stop();

    EXPECT_TRUE(handle.deviceData());// grid only exists on the GPU
    EXPECT_TRUE(handle.deviceGrid<BuildT>());
    EXPECT_FALSE(handle.deviceGrid<int>(0));
    EXPECT_TRUE(handle.deviceGrid<BuildT>(0));
    EXPECT_FALSE(handle.deviceGrid<BuildT>(1));
    EXPECT_FALSE(handle.data());// no grid was yet allocated on the CPU

    //timer.start("Allocating and copying grid from GPU to CPU");
    auto *grid = handle.grid<BuildT>();// no grid on the CPU
    EXPECT_FALSE(grid);
    handle.deviceDownload();// creates a copy on the CPU
    EXPECT_TRUE(handle.deviceData());
    EXPECT_TRUE(handle.data());
    auto *data = handle.gridData();
    EXPECT_TRUE(data);
    grid = handle.grid<BuildT>();
    EXPECT_TRUE(grid);
    EXPECT_TRUE(grid->valueCount()>0);
    EXPECT_EQ(nanovdb::Vec3d(1.0), grid->voxelSize());

    //timer.restart("Parallel unit-testing on CPU");
    nanovdb::forEach(voxels,[&](const nanovdb::Range1D &r){
        auto acc = grid->getAccessor();
        for (size_t i=r.begin(); i!=r.end(); ++i) {
            const nanovdb::Coord &ijk = voxels[i];
            EXPECT_TRUE(acc.probeLeaf(ijk)!=nullptr);
            EXPECT_TRUE(acc.isActive(ijk));
            EXPECT_TRUE(acc.getValue(ijk) > 0u);
            const auto *leaf = acc.get<nanovdb::GetLeaf<BuildT>>(ijk);
            EXPECT_TRUE(leaf);
            const auto offset = leaf->CoordToOffset(ijk);
            EXPECT_EQ(ijk, leaf->offsetToGlobalCoord(offset));
        }
    });

    //timer.stop();
}// Large_CudaPointsToGrid_old

TEST(TestNanoVDBCUDA, mergeSplitGrids)
{
    size_t size1 = 0, size2 = 0;
    std::vector<nanovdb::GridHandle<>> handles1, handles2;
    std::vector<std::string> gridNames;
    nanovdb::CpuTimer timer("create 5 host grids");
    for (int radius = 100; radius<150; radius += 10) {
        gridNames.emplace_back("sphere_" + std::to_string(radius));
        handles1.emplace_back(nanovdb::createLevelSetSphere(radius,nanovdb::Vec3d(0),1,3,
                                                            nanovdb::Vec3d(0), gridNames.back()));
        EXPECT_FALSE(handles1.back().isPadded());
        size1 += handles1.back().size();
    }
    EXPECT_EQ(5u, gridNames.size());
    EXPECT_EQ(5u, handles1.size());
    timer.restart("create 5 host grids");
    for (int radius = 150; radius<200; radius += 10) {
        gridNames.emplace_back("sphere_" + std::to_string(radius));
        handles2.emplace_back(nanovdb::createLevelSetSphere(radius,nanovdb::Vec3d(0),1,3,
                                                            nanovdb::Vec3d(0), gridNames.back()));
        size2 += handles2.back().size();
    }
    EXPECT_EQ(10u, gridNames.size());
    EXPECT_EQ( 5u, handles2.size());
    timer.restart("merging 5 host grids");
    auto mergedHandle = nanovdb::mergeGrids<nanovdb::HostBuffer, std::vector>(handles2);// merge last 5 grid handles
    EXPECT_EQ(size2, mergedHandle.size());
    EXPECT_FALSE(mergedHandle.isPadded());
    EXPECT_TRUE(mergedHandle.data());
    auto *gridData = mergedHandle.gridData();// first grid
    EXPECT_TRUE(gridData);
    EXPECT_EQ(5u, gridData->mGridCount);
    EXPECT_EQ(0u, gridData->mGridIndex);
    EXPECT_EQ(handles2[0].size(), gridData->mGridSize);
    timer.restart("unit-test host grids");
    for (int i=0; i<5; ++i){
        gridData = mergedHandle.gridData(i);
        EXPECT_TRUE(gridData);
        EXPECT_EQ(i, gridData->mGridIndex);
        EXPECT_EQ(handles2[i].size(), gridData->mGridSize);
        EXPECT_EQ(strcmp(gridNames[i+5].c_str(), gridData->mGridName),0);
    }

    EXPECT_FALSE(mergedHandle.empty());
    handles1.push_back(std::move(mergedHandle));// append one handle with 5 merged grids
    EXPECT_TRUE(mergedHandle.empty());
    EXPECT_EQ(6u, handles1.size());
    timer.restart("merging 10 host grids");
    mergedHandle = nanovdb::mergeGrids<nanovdb::HostBuffer, std::vector>(handles1);
    EXPECT_EQ(size1 + size2, mergedHandle.size());
    EXPECT_TRUE(mergedHandle.data());
    gridData = mergedHandle.gridData();// first grid
    EXPECT_TRUE(gridData);
    EXPECT_EQ(10u, gridData->mGridCount);
    EXPECT_EQ( 0u, gridData->mGridIndex);
    EXPECT_EQ(handles1[0].size(), gridData->mGridSize);

    timer.restart("splitting host grids");
    auto splitHandles = nanovdb::splitGrids(mergedHandle);
    timer.restart("unit-test split grids");
    EXPECT_EQ(10u, splitHandles.size());
    for (int i=0; i<5; ++i){
        EXPECT_EQ(handles1[i].size(), splitHandles[i].size());
        gridData = splitHandles[i].gridData();
        EXPECT_EQ(0u, gridData->mGridIndex);
        EXPECT_EQ(1u, gridData->mGridCount);
        EXPECT_EQ(strcmp(gridNames[i].c_str(), gridData->mGridName),0);
    }
    for (int i=5; i<10; ++i){
        EXPECT_EQ(handles2[i-5].size(), splitHandles[i].size());
        gridData = splitHandles[i].gridData();
        EXPECT_EQ(0u, gridData->mGridIndex);
        EXPECT_EQ(1u, gridData->mGridCount);
        EXPECT_EQ(strcmp(gridNames[i].c_str(), gridData->mGridName),0);
    }
    timer.stop();
}//  mergeSplitGrids

TEST(TestNanoVDBCUDA, mergeSplitDeviceGrids)
{
    using BufferT = nanovdb::CudaDeviceBuffer;
    using HandleT = nanovdb::GridHandle<BufferT>;
    size_t size = 0;
    std::vector<HandleT> handles;
    std::vector<std::string> gridNames;
    nanovdb::CpuTimer timer("create 10 host grids");
    for (int radius = 100; radius<200; radius += 10) {
        gridNames.emplace_back("sphere_" + std::to_string(radius));
        handles.emplace_back(nanovdb::createLevelSetSphere<float, BufferT>(radius,nanovdb::Vec3d(0),1,3,
                                                           nanovdb::Vec3d(0), gridNames.back()));
        EXPECT_FALSE(handles.back().isPadded());
        size += handles.back().size();
    }
    timer.restart("copy grids to device");
    for (auto &h : handles) h.deviceUpload();
    EXPECT_EQ(10u, handles.size());
    timer.restart("merging device grids");
    auto mergedHandle = nanovdb::mergeDeviceGrids<BufferT, std::vector>(handles);
    EXPECT_EQ(size, mergedHandle.size());
    EXPECT_FALSE(mergedHandle.data());
    EXPECT_TRUE(mergedHandle.deviceData());
    EXPECT_FALSE(mergedHandle.isPadded());
    timer.restart("copy grids to host");
    mergedHandle.deviceDownload();
    EXPECT_TRUE(mergedHandle.data());
    EXPECT_TRUE(mergedHandle.deviceData());
    EXPECT_FALSE(mergedHandle.isPadded());
    auto *gridData = mergedHandle.gridData();// first grid
    EXPECT_TRUE(gridData);
    EXPECT_EQ(10u, gridData->mGridCount);
    EXPECT_EQ(0u, gridData->mGridIndex);
    timer.restart("unit-test host grids");
    for (uint32_t i=0; i<10; ++i) {
        gridData = mergedHandle.gridData(i);
        EXPECT_TRUE(gridData);
        EXPECT_EQ(i, gridData->mGridIndex);
        EXPECT_EQ(strcmp(gridNames[i].c_str(), gridData->mGridName),0);
    }
    timer.restart("splitting device grids");
    auto splitHandles = nanovdb::splitDeviceGrids<BufferT, std::vector>(mergedHandle);
    timer.restart("unit-test split grids");
    EXPECT_EQ(10u, splitHandles.size());
    for (uint32_t i=0u; i<10u; ++i) {
        EXPECT_EQ(handles[i].size(), splitHandles[i].size());
        EXPECT_FALSE(splitHandles[i].isPadded());
        EXPECT_FALSE(splitHandles[i].gridData());
        splitHandles[i].deviceDownload();
        gridData = splitHandles[i].gridData();
        EXPECT_TRUE(gridData);
        EXPECT_EQ(0u, gridData->mGridIndex);
        EXPECT_EQ(1u, gridData->mGridCount);
        EXPECT_EQ(strcmp(gridNames[i].c_str(), gridData->mGridName),0);
    }
    timer.stop();
}//  mergeSplitDeviceGrids

// make -j 4 testNanoVDB && ./unittest/testNanoVDB --gtest_filter="*Cuda*" --gtest_break_on_failure
TEST(TestNanoVDBCUDA, CudaIndexGridToGrid_basic)
{
    using BufferT = nanovdb::CudaDeviceBuffer;
    const float value = 1.23456f, backgroud = 1.0f;
    const nanovdb::Coord ijk(1,2,3);
    nanovdb::GridHandle<BufferT> floatHdl;
    nanovdb::FloatGrid *floatGrid = nullptr;
    //nanovdb::CpuTimer timer;
    {// create float grid with one active voxel
        nanovdb::build::Grid<float> grid(backgroud);
        auto srcAcc = grid.getAccessor();
        srcAcc.setValue(ijk, value);
        auto nodeCount = grid.nodeCount();
        EXPECT_EQ(1u, nodeCount[0]);
        EXPECT_EQ(1u, nodeCount[1]);
        EXPECT_EQ(1u, nodeCount[2]);
        EXPECT_EQ(value, srcAcc.getValue(ijk));
        EXPECT_EQ(value, srcAcc.getValue(1,2,3));
        //timer.start("Create FloatGrid on CPU");
        floatHdl = nanovdb::createNanoGrid<nanovdb::build::Grid<float>, float, BufferT>(grid);
        EXPECT_TRUE(floatHdl);
        floatGrid = floatHdl.grid<float>();
        EXPECT_TRUE(floatGrid);
        EXPECT_EQ(ijk, floatGrid->indexBBox()[0]);
        EXPECT_EQ(ijk, floatGrid->indexBBox()[1]);
        auto acc = floatGrid->getAccessor();
        EXPECT_EQ(backgroud, acc.getValue(nanovdb::Coord(-1)));
        EXPECT_FALSE(acc.isActive(nanovdb::Coord(-1)));
        EXPECT_EQ(backgroud, acc.getValue(nanovdb::Coord(8)));
        EXPECT_FALSE(acc.isActive(nanovdb::Coord(8)));
        EXPECT_EQ(backgroud, acc.getValue(nanovdb::Coord(0)));
        EXPECT_FALSE(acc.isActive(nanovdb::Coord(0)));
        EXPECT_EQ(value, acc.getValue(ijk));
        EXPECT_TRUE(acc.isActive(ijk));
    }
    //timer.restart("Create IndexGrid on CPU");
    using BufferT = nanovdb::CudaDeviceBuffer;
    auto idxHdl = nanovdb::createNanoGrid<nanovdb::FloatGrid, nanovdb::ValueIndex, BufferT>(*floatGrid, 0u, false, false, 1);
    //timer.restart("Copy IndexGrid from CPU to GPU");
    EXPECT_FALSE(idxHdl.deviceGrid<nanovdb::ValueIndex>());
    idxHdl.deviceUpload();
    EXPECT_TRUE(idxHdl.deviceGrid<nanovdb::ValueIndex>());
    auto *idxGrid = idxHdl.grid<nanovdb::ValueIndex>();
    EXPECT_TRUE(idxGrid);
    //timer.restart("Create value list on CPU");
    EXPECT_EQ(1u + 512u, idxGrid->valueCount());// background + 512 values in one leaf node
    float *values = new float[idxGrid->valueCount()], *d_values = nullptr;
    values[0] = backgroud;
    const float *q = floatGrid->tree().getFirstLeaf()->data()->mValues;
    for (float *p=values+1, *e=p+512;p!=e; ++p) *p = *q++;
    //timer.restart("Allocate and copy values from CPU to GPU");
    cudaCheck(hipMalloc((void**)&d_values, idxGrid->valueCount()*sizeof(float)));
    EXPECT_TRUE(d_values);
    cudaCheck(hipMemcpy(d_values, values, idxGrid->valueCount()*sizeof(float), hipMemcpyHostToDevice));
    EXPECT_FALSE(idxHdl.deviceGrid<float>());
    auto *d_idxGrid = idxHdl.deviceGrid<nanovdb::ValueIndex>();
    EXPECT_TRUE(d_idxGrid);
    //timer.restart("Call CudaIndexToGrid");
    auto hdl = nanovdb::cudaIndexToGrid<float>(d_idxGrid, d_values);
    //timer.restart("unit-test");
    EXPECT_FALSE(hdl.grid<float>());// no host grid
    EXPECT_TRUE(hdl.deviceGrid<float>());
    hdl.deviceDownload();
    auto *floatGrid2 = hdl.grid<float>();
    EXPECT_TRUE(floatGrid2);
    auto *leaf2 = floatGrid2->tree().getFirstLeaf();
    EXPECT_TRUE(leaf2);
    auto acc  = floatGrid->getAccessor();
    auto acc2 = floatGrid2->getAccessor();
    EXPECT_EQ(floatGrid->indexBBox(), floatGrid2->indexBBox());
    EXPECT_EQ(floatGrid->worldBBox(), floatGrid2->worldBBox());
    // probe background in root node
    EXPECT_EQ(backgroud, acc.getValue(nanovdb::Coord(-1)));
    EXPECT_FALSE(acc.isActive(nanovdb::Coord(-1)));
    EXPECT_EQ(backgroud, acc2.getValue(nanovdb::Coord(-1)));
    EXPECT_FALSE(acc2.isActive(nanovdb::Coord(-1)));
    // probe background in upper node
    EXPECT_EQ(backgroud, acc.getValue(nanovdb::Coord(128)));
    EXPECT_FALSE(acc.isActive(nanovdb::Coord(128)));
    EXPECT_EQ(backgroud, floatGrid2->tree().getValue(nanovdb::Coord(128)));
    EXPECT_EQ(backgroud, acc2.getValue(nanovdb::Coord(128)));
    EXPECT_FALSE(acc2.isActive(nanovdb::Coord(128)));
    // probe background in leaf node
    EXPECT_EQ(backgroud, acc.getValue(nanovdb::Coord(0)));
    EXPECT_FALSE(acc.isActive(nanovdb::Coord(0)));
    EXPECT_EQ(backgroud, leaf2->getValue(nanovdb::Coord(0)));
    EXPECT_FALSE(leaf2->isActive(nanovdb::Coord(0)));
    EXPECT_EQ(backgroud, floatGrid2->tree().getValue(nanovdb::Coord(0)));
    EXPECT_EQ(backgroud, acc2.getValue(nanovdb::Coord(0)));
    EXPECT_FALSE(acc2.isActive(nanovdb::Coord(0)));

    EXPECT_EQ(value, acc2.getValue(ijk));
    EXPECT_TRUE(acc2.isActive(ijk));
    //timer.stop();
    hipFree(d_values);
}//  CudaIndexGridToGrid_basic

TEST(TestNanoVDBCUDA, CudaIndexGridToGrid_ValueIndex)
{
    using BuildT = nanovdb::ValueIndex;
    using BufferT = nanovdb::CudaDeviceBuffer;
    //nanovdb::CpuTimer timer("Create FloatGrid on CPU");
    auto floatHdl = nanovdb::createLevelSetSphere<float, BufferT>(100,nanovdb::Vec3d(0),1,3, nanovdb::Vec3d(0), "test");
    auto *floatGrid = floatHdl.grid<float>();
    EXPECT_TRUE(floatGrid);
    auto acc = floatGrid->getAccessor();
    //timer.restart("Create IndexGrid on CPU");
    auto idxHdl = nanovdb::createNanoGrid<nanovdb::FloatGrid, BuildT, BufferT>(*floatGrid);
    //timer.restart("Copy IndexGrid from CPU to GPU");
    idxHdl.deviceUpload();
    auto *idxGrid = idxHdl.grid<BuildT>();
    EXPECT_TRUE(idxGrid);
    //timer.restart("Create value list on CPU");
    float *values = new float[idxGrid->valueCount()], *d_values = nullptr;
    values[0] = floatGrid->tree().root().background();
    for (auto it = floatGrid->indexBBox().begin(); it; ++it) {
        EXPECT_EQ(acc.isActive(*it), idxGrid->tree().isActive(*it));
        const uint64_t idx = idxGrid->tree().getValue(*it);
        EXPECT_TRUE(idx < idxGrid->valueCount());
        values[idx] = acc.getValue(*it);
    }
    //timer.restart("Allocate and copy values from CPU to GPU");
    cudaCheck(hipMalloc((void**)&d_values, idxGrid->valueCount()*sizeof(float)));
    cudaCheck(hipMemcpy(d_values, values, idxGrid->valueCount()*sizeof(float), hipMemcpyHostToDevice));
    EXPECT_FALSE(idxHdl.deviceGrid<float>());
    auto *d_idxGrid = idxHdl.deviceGrid<BuildT>();
    EXPECT_TRUE(d_idxGrid);
    //timer.restart("Call CudaIndexToGrid");
    auto hdl = nanovdb::cudaIndexToGrid<float>(d_idxGrid, d_values);
    //timer.restart("unit-test");
    EXPECT_FALSE(hdl.grid<float>());// no host grid
    EXPECT_TRUE(hdl.deviceGrid<float>());
    hdl.deviceDownload();
    auto *floatGrid2 = hdl.grid<float>();
    EXPECT_TRUE(floatGrid2);
    auto acc2 = floatGrid2->getAccessor();
    EXPECT_EQ(floatGrid->indexBBox(), floatGrid2->indexBBox());
    EXPECT_EQ(floatGrid->worldBBox(), floatGrid2->worldBBox());
    EXPECT_EQ(floatGrid->tree().root().background(), floatGrid2->tree().root().background());
    for (auto it = floatGrid->indexBBox().begin(); it; ++it) {
        EXPECT_EQ(acc.isActive(*it), acc2.isActive(*it));
        EXPECT_EQ(acc.getValue(*it), acc2.getValue(*it));
    }
    //timer.stop();
    hipFree(d_values);
}//  CudaPointToGrid_ValueIndex

TEST(TestNanoVDBCUDA, CudaIndexGridToGrid_ValueOnIndex)
{
    using BuildT = nanovdb::ValueOnIndex;
    using BufferT = nanovdb::CudaDeviceBuffer;
    //nanovdb::CpuTimer timer("Create FloatGrid on CPU");
    auto floatHdl = nanovdb::createLevelSetSphere<float, BufferT>(100,nanovdb::Vec3d(0),1,3, nanovdb::Vec3d(0), "test");
    auto *floatGrid = floatHdl.grid<float>();
    EXPECT_TRUE(floatGrid);
    auto acc = floatGrid->getAccessor();
    //timer.restart("Create IndexGrid on CPU");
    auto idxHdl = nanovdb::createNanoGrid<nanovdb::FloatGrid, BuildT, BufferT>(*floatGrid);
    //timer.restart("Copy IndexGrid from CPU to GPU");
    idxHdl.deviceUpload();
    auto *idxGrid = idxHdl.grid<BuildT>();
    EXPECT_TRUE(idxGrid);
    //timer.restart("Create value list on CPU");
    float *values = new float[idxGrid->valueCount()], *d_values = nullptr;
    values[0] = floatGrid->tree().root().background();
    for (auto it = floatGrid->indexBBox().begin(); it; ++it) {
        EXPECT_EQ(acc.isActive(*it), idxGrid->tree().isActive(*it));
        if (acc.isActive(*it)) {
            const uint64_t idx = idxGrid->tree().getValue(*it);
            EXPECT_TRUE(idx < idxGrid->valueCount());
            values[idx] = acc.getValue(*it);
        }
    }
    //timer.restart("Allocate and copy values from CPU to GPU");
    cudaCheck(hipMalloc((void**)&d_values, idxGrid->valueCount()*sizeof(float)));
    cudaCheck(hipMemcpy(d_values, values, idxGrid->valueCount()*sizeof(float), hipMemcpyHostToDevice));
    EXPECT_FALSE(idxHdl.deviceGrid<float>());
    auto *d_idxGrid = idxHdl.deviceGrid<BuildT>();
    EXPECT_TRUE(d_idxGrid);
    //timer.restart("Call CudaIndexToGrid");
    auto hdl = nanovdb::cudaIndexToGrid<float>(d_idxGrid, d_values);
    //timer.restart("unit-test");
    EXPECT_FALSE(hdl.grid<float>());// no host grid
    EXPECT_TRUE(hdl.deviceGrid<float>());
    hdl.deviceDownload();
    auto *floatGrid2 = hdl.grid<float>();
    EXPECT_TRUE(floatGrid2);
    auto acc2 = floatGrid2->getAccessor();
    EXPECT_EQ(floatGrid->indexBBox(), floatGrid2->indexBBox());
    EXPECT_EQ(floatGrid->worldBBox(), floatGrid2->worldBBox());
    EXPECT_EQ(floatGrid->tree().root().background(), floatGrid2->tree().root().background());
    for (auto it = floatGrid->indexBBox().begin(); it; ++it) {
        EXPECT_EQ(acc.isActive(*it), acc2.isActive(*it));
        if (acc.isActive(*it)) EXPECT_EQ(acc.getValue(*it), acc2.getValue(*it));
    }
    //timer.stop();
    hipFree(d_values);
}//  CudaPointToGrid_ValueOnIndex

TEST(TestNanoVDBCUDA, CudaSignedFloodFill)
{
    using BufferT = nanovdb::CudaDeviceBuffer;
    //nanovdb::CpuTimer timer("Create FloatGrid on CPU");
    auto floatHdl = nanovdb::createLevelSetSphere<float, BufferT>(100);
    auto *floatGrid = floatHdl.grid<float>();
    EXPECT_TRUE(floatGrid);
    auto acc = floatGrid->getAccessor();
    EXPECT_FALSE(acc.isActive(nanovdb::Coord(103,0,0)));
    EXPECT_TRUE( acc.isActive(nanovdb::Coord(100,0,0)));
    EXPECT_FALSE(acc.isActive(nanovdb::Coord( 97,0,0)));
    EXPECT_EQ( 3.0f, acc(103,0,0));
    EXPECT_EQ( 0.0f, acc(100,0,0));
    EXPECT_EQ(-3.0f, acc( 97,0,0));
    using OpT = nanovdb::SetVoxel<float>;// only set the voxel value
    acc.set<OpT>(nanovdb::Coord(103,0,0),-1.0f);// flip sign and value of inactive voxel
    acc.set<OpT>(nanovdb::Coord( 97,0,0), 1.0f);// flip sign and value of inactive voxel
    EXPECT_EQ(-1.0f, acc(103,0,0));
    EXPECT_EQ( 0.0f, acc(100,0,0));
    EXPECT_EQ( 1.0f, acc( 97,0,0));
    //timer.restart("Copy FloatGrid from CPU to GPU");
    floatHdl.deviceUpload();// CPU -> GPU
    auto *d_floatGrid = floatHdl.deviceGrid<float>();
    EXPECT_TRUE(d_floatGrid);
    //timer.restart("Signed flood-fill on the GPU");
    //nanovdb::cudaSignedFloodFill(d_floatGrid, true);
    nanovdb::cudaSignedFloodFill(d_floatGrid);
    //timer.restart("Copy FloatGrid from GPU to CPU");
    floatHdl.deviceDownload();// GPU -> CPU
    //timer.stop();
    floatGrid = floatHdl.grid<float>();
    EXPECT_TRUE(floatGrid);
    acc = floatGrid->getAccessor();
    EXPECT_EQ( 3.0f, acc(103,0,0));
    EXPECT_EQ( 0.0f, acc(100,0,0));
    EXPECT_EQ(-3.0f, acc( 97,0,0));
}//  CudaSignedFloodFill

TEST(TestNanoVDBCUDA, OneVoxelToGrid)
{
    using BuildT = float;
    using GridT = nanovdb::NanoGrid<BuildT>;
    const size_t num_points = 1;
    nanovdb::Coord coords[num_points] = {nanovdb::Coord(1, 2, 3)}, *d_coords = nullptr;
    cudaCheck(hipMalloc(&d_coords, num_points * sizeof(nanovdb::Coord)));
    cudaCheck(hipMemcpy(d_coords, coords, num_points * sizeof(nanovdb::Coord), hipMemcpyHostToDevice));// CPU -> GPU

    //nanovdb::GpuTimer timer("Create FloatGrid on GPU");
    nanovdb::CudaPointsToGrid<BuildT> converter;
    auto handle = converter.getHandle(d_coords, num_points);
    cudaCheck(hipFree(d_coords));
    //timer.stop();

    EXPECT_TRUE(handle.deviceData());// grid only exists on the GPU
    EXPECT_FALSE(handle.data());// no grid was yet allocated on the CPU

    const uint64_t size = sizeof(GridT) +
                          sizeof(GridT::TreeType) +
                          GridT::RootType::memUsage(1) +
                          sizeof(GridT::UpperNodeType) +
                          sizeof(GridT::LowerNodeType) +
                          sizeof(GridT::LeafNodeType);
    EXPECT_EQ(handle.size(), size);

    GridT *grid = handle.grid<BuildT>();// no grid on the CPU
    EXPECT_FALSE(grid);
    //timer.start("Copy data from GPU to CPU");
    handle.deviceDownload();// creates a copy up the CPU
    //timer.stop();
    EXPECT_TRUE(handle.deviceData());
    EXPECT_TRUE(handle.data());
    auto *data = handle.gridData();
    EXPECT_TRUE(data);
    grid = handle.grid<BuildT>();
    EXPECT_TRUE(grid);

    //timer.start("Unit-testing grid on the CPU");
    auto acc = grid->getAccessor();
    EXPECT_FALSE(acc.isActive(nanovdb::Coord(0,2,3)));
    EXPECT_TRUE( acc.isActive(nanovdb::Coord(1,2,3)));
    EXPECT_FALSE(acc.isActive(nanovdb::Coord(1,2,4)));
    EXPECT_FALSE(acc.isActive(nanovdb::Coord(2,2,3)));
    auto *leaf = acc.probeLeaf(nanovdb::Coord(1,2,3));
    EXPECT_TRUE(leaf);
    EXPECT_EQ(nanovdb::Coord(0), leaf->origin());
    EXPECT_EQ(1u, leaf->valueMask().countOn());
    EXPECT_EQ(nanovdb::Coord(1,2,3), leaf->bbox()[0]);
    EXPECT_EQ(nanovdb::Coord(1,2,3), leaf->bbox()[1]);
    auto *lower = acc.getNode<1>();
    EXPECT_TRUE(lower);
    EXPECT_EQ(nanovdb::Coord(1,2,3), lower->bbox()[0]);
    EXPECT_EQ(nanovdb::Coord(1,2,3), lower->bbox()[1]);
    auto *upper = acc.getNode<2>();
    EXPECT_TRUE(upper);
    EXPECT_EQ(nanovdb::Coord(1,2,3), upper->bbox()[0]);
    EXPECT_EQ(nanovdb::Coord(1,2,3), upper->bbox()[1]);
    EXPECT_EQ(nanovdb::Coord(1,2,3), acc.root().bbox()[0]);
    EXPECT_EQ(nanovdb::Coord(1,2,3), acc.root().bbox()[1]);
    //timer.stop();
}// OneVoxelToGrid

TEST(TestNanoVDBCUDA, ThreePointsToGrid)
{
    using BuildT = nanovdb::Points;
    using Vec3T  = nanovdb::Vec3f;
    using GridT  = nanovdb::NanoGrid<BuildT>;
    const size_t num_points = 3;
    Vec3T points[num_points] = {Vec3T(1, 0, 0),Vec3T(1, 2, 3),Vec3T(1, 2, 3)}, *d_points = nullptr;
    cudaCheck(hipMalloc(&d_points, num_points * sizeof(Vec3T)));
    cudaCheck(hipMemcpy(d_points, points, num_points * sizeof(Vec3T), hipMemcpyHostToDevice));// CPU -> GPU

    //nanovdb::GpuTimer timer("Create FloatGrid on GPU");
    nanovdb::CudaPointsToGrid<BuildT> converter;
    auto handle = converter.getHandle(d_points, num_points);
    cudaCheck(hipFree(d_points));
    //timer.stop();

    EXPECT_TRUE(handle.deviceData());// grid only exists on the GPU
    EXPECT_FALSE(handle.data());// no grid was yet allocated on the CPU

    const uint64_t size = sizeof(GridT) +
                          sizeof(GridT::TreeType) +
                          GridT::RootType::memUsage(1) +
                          sizeof(GridT::UpperNodeType) +
                          sizeof(GridT::LowerNodeType) +
                          sizeof(GridT::LeafNodeType) +
                          sizeof(nanovdb::GridBlindMetaData) +
                          num_points*sizeof(Vec3T);
    EXPECT_EQ(handle.size(), size);

    GridT *grid = handle.grid<BuildT>();// no grid on the CPU
    EXPECT_FALSE(grid);
    //timer.start("Copy data from GPU to CPU");
    handle.deviceDownload();// creates a copy on the CPU
    //timer.stop();
    EXPECT_TRUE(handle.deviceData());
    EXPECT_TRUE(handle.data());
    auto *data = handle.gridData();
    EXPECT_TRUE(data);
    grid = handle.grid<BuildT>();
    EXPECT_TRUE(grid);
    EXPECT_EQ(1u, grid->blindDataCount());
    const Vec3T *blindData = grid->getBlindData<Vec3T>(0);
    EXPECT_TRUE(blindData);
    for (const Vec3T *p = blindData, *q=p+num_points, *ptr=points; p!=q; ++p) {
        EXPECT_EQ(*ptr++, *p);
    }
    //timer.start("Unit-testing grid on the CPU");
    nanovdb::PointAccessor<Vec3T, BuildT> acc(*grid);
    EXPECT_TRUE(acc);
    EXPECT_FALSE(acc.isActive(nanovdb::Coord(0,2,3)));
    EXPECT_TRUE( acc.isActive(nanovdb::Coord(1,0,0)));
    EXPECT_TRUE( acc.isActive(nanovdb::Coord(1,2,3)));
    EXPECT_FALSE(acc.isActive(nanovdb::Coord(1,2,4)));
    EXPECT_FALSE(acc.isActive(nanovdb::Coord(2,2,3)));
    auto *leaf = acc.probeLeaf(nanovdb::Coord(1,2,3));
    EXPECT_TRUE(leaf);
    EXPECT_EQ(nanovdb::Coord(0), leaf->origin());
    EXPECT_EQ(2u, leaf->valueMask().countOn());
    EXPECT_EQ(nanovdb::Coord(1,0,0), leaf->bbox()[0]);
    EXPECT_EQ(nanovdb::Coord(1,2,3), leaf->bbox()[1]);
    nanovdb::CoordBBox bbox(nanovdb::Coord(0), nanovdb::Coord(7));
    for (auto it = bbox.begin(); it; ++it) {
        //std::cerr << *it << " offset = " << leaf->CoordToOffset(*it) << " value = " << leaf->getValue(*it) << std::endl;
        if (*it < nanovdb::Coord(1,0,0)) {
            EXPECT_EQ(0u, leaf->getValue(*it));
        } else if (*it < nanovdb::Coord(1,2,3)) {
            EXPECT_EQ(1u, leaf->getValue(*it));
        } else {
            EXPECT_EQ(3u, leaf->getValue(*it));
        }
    }
    const Vec3T *start=nullptr, *stop=nullptr;

    EXPECT_EQ(0u, acc.voxelPoints(nanovdb::Coord(0,0,0), start, stop));
    EXPECT_FALSE(start);
    EXPECT_FALSE(stop);

    EXPECT_EQ(1u, acc.voxelPoints(nanovdb::Coord(1,0,0), start, stop));
    EXPECT_TRUE(start);
    EXPECT_TRUE(stop);
    EXPECT_LT(start, stop);
    EXPECT_EQ(Vec3T(1, 0, 0), start[0]);

    EXPECT_EQ(2u, acc.voxelPoints(nanovdb::Coord(1,2,3), start, stop));
    EXPECT_TRUE(start);
    EXPECT_TRUE(stop);
    EXPECT_LT(start, stop);
    EXPECT_EQ(Vec3T(1, 2, 3), start[0]);
    EXPECT_EQ(Vec3T(1, 2, 3), start[1]);

    auto *lower = acc.getNode<1>();
    EXPECT_TRUE(lower);
    EXPECT_EQ(nanovdb::Coord(1,0,0), lower->bbox()[0]);
    EXPECT_EQ(nanovdb::Coord(1,2,3), lower->bbox()[1]);
    auto *upper = acc.getNode<2>();
    EXPECT_TRUE(upper);
    EXPECT_EQ(nanovdb::Coord(1,0,0), upper->bbox()[0]);
    EXPECT_EQ(nanovdb::Coord(1,2,3), upper->bbox()[1]);
    EXPECT_EQ(nanovdb::Coord(1,0,0), acc.root().bbox()[0]);
    EXPECT_EQ(nanovdb::Coord(1,2,3), acc.root().bbox()[1]);
    //timer.stop();
}// ThreePointsToGrid

TEST(TestNanoVDBCUDA, EightVoxelsToFloatGrid)
{
    using BuildT = float;
    using GridT = nanovdb::NanoGrid<BuildT>;
    const size_t num_points = 8;
    //std::cerr << nanovdb::NanoLeaf<uint32_t>::CoordToOffset(nanovdb::Coord( 1, 1, 1)) << std::endl;
    //std::cerr << nanovdb::NanoLeaf<uint32_t>::CoordToOffset(nanovdb::Coord(-7, 1, 1)) << std::endl;
    //std::cerr << nanovdb::NanoLeaf<uint32_t>::CoordToOffset(nanovdb::Coord( 1,-7, 1)) << std::endl;
    //std::cerr << nanovdb::NanoLeaf<uint32_t>::CoordToOffset(nanovdb::Coord( 1,-7, 1)) << std::endl;
    nanovdb::Coord coords[num_points] = {nanovdb::Coord( 1, 1, 1),
                                         nanovdb::Coord(-7, 1, 1),
                                         nanovdb::Coord( 1,-7, 1),
                                         nanovdb::Coord( 1, 1,-7),
                                         nanovdb::Coord(-7,-7, 1),
                                         nanovdb::Coord(-7, 1,-7),
                                         nanovdb::Coord( 1,-7,-7),
                                         nanovdb::Coord(-7,-7,-7)}, *d_coords = nullptr;
    for (int i=0; i<8; ++i) EXPECT_EQ(73u, nanovdb::NanoLeaf<uint32_t>::CoordToOffset(coords[i]));
    cudaCheck(hipMalloc(&d_coords, num_points * sizeof(nanovdb::Coord)));
    cudaCheck(hipMemcpy(d_coords, coords, num_points * sizeof(nanovdb::Coord), hipMemcpyHostToDevice));// CPU -> GPU

    //nanovdb::GpuTimer timer("Create FloatGrid on GPU");
    nanovdb::CudaPointsToGrid<BuildT> converter;
    auto handle = converter.getHandle(d_coords, num_points);
    //timer.stop();
    cudaCheck(hipFree(d_coords));

    EXPECT_TRUE(handle.deviceData());// grid only exists on the GPU
    EXPECT_FALSE(handle.data());// no grid was yet allocated on the CPU

    const uint64_t size = sizeof(GridT) +
                          sizeof(GridT::TreeType) +
                          GridT::RootType::memUsage(8) +
                          8*sizeof(GridT::UpperNodeType) +
                          8*sizeof(GridT::LowerNodeType) +
                          8*sizeof(GridT::LeafNodeType);
    EXPECT_EQ(handle.size(), size);

    GridT *grid = handle.grid<BuildT>();// no grid on the CPU
    EXPECT_FALSE(grid);
    //timer.start("Copy data from GPU to CPU");
    handle.deviceDownload();// creates a copy up the CPU
    //timer.stop();
    EXPECT_TRUE(handle.deviceData());
    EXPECT_TRUE(handle.data());
    auto *data = handle.gridData();
    EXPECT_TRUE(data);
    grid = handle.grid<BuildT>();
    EXPECT_TRUE(grid);

    //timer.start("Unit-testing grid on the CPU");
    auto acc = grid->getAccessor();
    EXPECT_FALSE(acc.isActive(nanovdb::Coord(0,2,3)));
    EXPECT_TRUE( acc.isActive(nanovdb::Coord(1,1,1)));
    EXPECT_FALSE(acc.isActive(nanovdb::Coord(1,2,4)));
    EXPECT_FALSE(acc.isActive(nanovdb::Coord(2,2,3)));
    auto *leaf = acc.probeLeaf(nanovdb::Coord(1,0,0));
    EXPECT_TRUE(leaf);
    EXPECT_EQ(nanovdb::Coord(0), leaf->origin());
    EXPECT_EQ(1u, leaf->valueMask().countOn());
    EXPECT_EQ(nanovdb::Coord( 1, 1, 1), leaf->bbox()[0]);
    EXPECT_EQ(nanovdb::Coord( 1, 1, 1), leaf->bbox()[1]);
    auto *lower = acc.getNode<1>();
    EXPECT_TRUE(lower);
    EXPECT_EQ(nanovdb::Coord(1,1,1), lower->bbox()[0]);
    EXPECT_EQ(nanovdb::Coord(1,1,1), lower->bbox()[1]);
    auto *upper = acc.getNode<2>();
    EXPECT_TRUE(upper);
    EXPECT_EQ(nanovdb::Coord(1,1,1), upper->bbox()[0]);
    EXPECT_EQ(nanovdb::Coord(1,1,1), upper->bbox()[1]);
    EXPECT_EQ(nanovdb::Coord(-7,-7,-7), acc.root().bbox()[0]);
    EXPECT_EQ(nanovdb::Coord( 1, 1, 1), acc.root().bbox()[1]);
    //timer.stop();
}// EightVoxelsToFloatGrid

TEST(TestNanoVDBCUDA, Random_CudaPointsToGrid_World64)
{
    using BuildT = nanovdb::Points;//uint32_t;
    using Vec3T = nanovdb::Vec3d;
    //nanovdb::CpuTimer timer;
    const size_t pointCount = 1 << 20;// 1048576
    std::vector<Vec3T> points;
    //generate random points
    points.reserve(pointCount);
    std::srand(98765);
    const int max = 512, min = -max;
    auto op = [&](){return rand() % (max - min) + min;};
    //timer.start("Creating "+std::to_string(pointCount)+" random points on the CPU");
    while (points.size() < pointCount) points.emplace_back(op(), op(), op());
    //timer.stop();
    EXPECT_EQ(pointCount, points.size());
    Vec3T* d_points;
    const size_t pointSize = points.size() * sizeof(Vec3T);
    //std::cerr << "Point footprint: " << (pointSize >> 20) << " MB" << std::endl;
    //timer.start("Allocating "+std::to_string(pointSize >> 20)+" MB on the GPU");
    cudaCheck(hipMalloc(&d_points, pointSize));
    //timer.restart("Copying points from CPU to GPU");
    cudaCheck(hipMemcpy(d_points, points.data(), pointSize, hipMemcpyHostToDevice));
    //timer.stop();

    const double voxelSize = 8.0;
    //timer.start("Building grid on GPU from "+std::to_string(points.size())+" points");
    nanovdb::CudaPointsToGrid<BuildT> converter(voxelSize);// unit map
    //converter.setVerbose();
    auto handle = converter.getHandle(d_points, pointCount);
    //timer.stop();
    cudaCheck(hipFree(d_points));
    //std::cerr << "Grid size: " << (handle.size() >> 20) << " MB" << std::endl;

    const uint32_t maxPointsPerVoxel = converter.maxPointsPerVoxel();
    const uint32_t maxPointsPerLeaf  = converter.maxPointsPerLeaf();
    EXPECT_GT(maxPointsPerVoxel, 0u);
    EXPECT_LT(maxPointsPerLeaf, 1024u);
    EXPECT_LE(maxPointsPerVoxel, maxPointsPerLeaf);
    //std::cerr << "maxPointsPerLeaf = " << maxPointsPerLeaf << " maxPointsPerVoxel = " << maxPointsPerVoxel << std::endl;

    EXPECT_TRUE(handle.deviceData());// grid only exists on the GPU
    EXPECT_TRUE(handle.deviceGrid<BuildT>());
    EXPECT_FALSE(handle.deviceGrid<int>(0));
    EXPECT_TRUE(handle.deviceGrid<BuildT>(0));
    EXPECT_FALSE(handle.deviceGrid<BuildT>(1));
    EXPECT_FALSE(handle.data());// no grid was yet allocated on the CPU

    //timer.start("Allocating and copying grid from GPU to CPU");
    auto *grid = handle.grid<BuildT>();// no grid on the CPU
    EXPECT_FALSE(grid);
    handle.deviceDownload();// creates a copy on the CPU
    EXPECT_TRUE(handle.deviceData());
    EXPECT_TRUE(handle.data());
    auto *data = handle.gridData();
    EXPECT_TRUE(data);
    grid = handle.grid<BuildT>();
    EXPECT_TRUE(grid);
    EXPECT_EQ(nanovdb::Vec3d(voxelSize), grid->voxelSize());
    EXPECT_TRUE(nanovdb::CoordBBox::createCube(min, max-1).isInside(grid->indexBBox()));
    //std::cerr << grid->indexBBox() << std::endl;
    EXPECT_STREQ("World64: Vec3<double> point coordinates in world space", grid->blindMetaData(0).mName);
    {
        auto mgrHdl = nanovdb::createNodeManager(*grid);
        auto *mgr = mgrHdl.mgr<BuildT>();
        EXPECT_TRUE(mgr);
        for (uint32_t i=0; i<mgr->leafCount(); ++i) {
            const auto &leaf = mgr->leaf(i);
            for (int j=0; j<512; ++j) {
                EXPECT_LE(leaf.getValue(j), maxPointsPerLeaf);
                if (leaf.isActive(j)) {
                    if (j>0) {
                        EXPECT_LE(leaf.getValue(j) - leaf.getValue(j-1), maxPointsPerVoxel);
                    } else {
                        EXPECT_LE(leaf.getValue(0), maxPointsPerVoxel);
                    }
                } else if (j>0) {
                    EXPECT_EQ(leaf.getValue(j), leaf.getValue(j-1));
                } else {
                    EXPECT_EQ(leaf.getValue(0), 0u);
                }
            }// loop over voxels
        }// loop over leaf nodes
    }

    //timer.restart("Parallel unit-testing on CPU");
    nanovdb::forEach(points,[&](const nanovdb::Range1D &r){
        nanovdb::PointAccessor<Vec3T, BuildT> acc(*grid);
        EXPECT_TRUE(acc);
        const Vec3T *start = nullptr, *stop = nullptr;
        for (size_t i=r.begin(); i!=r.end(); ++i) {
            const nanovdb::Coord ijk = grid->worldToIndex(points[i]).round();
            EXPECT_TRUE(acc.probeLeaf(ijk)!=nullptr);
            EXPECT_TRUE(acc.isActive(ijk));
            EXPECT_LE(acc.getValue(ijk), pointCount);
            const auto *leaf = acc.get<nanovdb::GetLeaf<BuildT>>(ijk);
            EXPECT_TRUE(leaf);
            const auto offset = leaf->CoordToOffset(ijk);
            EXPECT_EQ(ijk, leaf->offsetToGlobalCoord(offset));
            const uint64_t count = acc.voxelPoints(ijk, start, stop);
            EXPECT_TRUE(start);
            EXPECT_TRUE(stop);
            EXPECT_LT(start, stop);
            EXPECT_LE(count, maxPointsPerVoxel);
            bool test = false;
            for (uint64_t j=0; test == false && j<count; ++j) test = (points[i] - start[j]).length() < 1e-9;
            EXPECT_TRUE(test);
        }
    });

    //timer.stop();
}// Random_CudaPointsToGrid_World64

TEST(TestNanoVDBCUDA, Large_CudaPointsToGrid_World64)
{
    using BuildT = nanovdb::Points;
    using Vec3T  = nanovdb::Vec3d;
    //nanovdb::CpuTimer timer;
    const size_t pointCount = 1 << 20;// 1048576
    std::vector<Vec3T> points;
    //generate random points
    points.reserve(pointCount);
    std::srand(98765);
    const int max = 512, min = -max;
    auto op = [&](){return rand() % (max - min) + min;};
    //timer.start("Creating "+std::to_string(pointCount)+" random points on the CPU");
    while (points.size() < pointCount) points.emplace_back(op(), op(), op());
    //timer.stop();
    EXPECT_EQ(pointCount, points.size());
    Vec3T* d_points;
    const size_t pointSize = points.size() * sizeof(Vec3T);
    //std::cerr << "Point footprint: " << (pointSize >> 20) << " MB" << std::endl;
    //timer.start("Allocating "+std::to_string(pointSize >> 20)+" MB on the GPU");
    cudaCheck(hipMalloc(&d_points, pointSize));
    //timer.restart("Copying points from CPU to GPU");
    cudaCheck(hipMemcpy(d_points, points.data(), pointSize, hipMemcpyHostToDevice));
    //timer.stop();

    const double voxelSize = 8.0;
    //timer.start("Building grid on GPU from "+std::to_string(points.size())+" points");
    nanovdb::CudaPointsToGrid<BuildT> converter(voxelSize);// unit map
    //converter.setVerbose();
    auto handle = converter.getHandle(d_points, pointCount);
    //timer.stop();
    cudaCheck(hipFree(d_points));
    //std::cerr << "Grid size: " << (handle.size() >> 20) << " MB" << std::endl;

    const uint32_t maxPointsPerVoxel = converter.maxPointsPerVoxel();
    const uint32_t maxPointsPerLeaf  = converter.maxPointsPerLeaf();
    EXPECT_GT(maxPointsPerVoxel, 0u);
    EXPECT_LT(maxPointsPerLeaf, 1024u);
    EXPECT_LE(maxPointsPerVoxel, maxPointsPerLeaf);
    //std::cerr << "maxPointsPerLeaf = " << maxPointsPerLeaf << " maxPointsPerVoxel = " << maxPointsPerVoxel << std::endl;

    EXPECT_TRUE(handle.deviceData());// grid only exists on the GPU
    EXPECT_TRUE(handle.deviceGrid<BuildT>());
    EXPECT_FALSE(handle.deviceGrid<int>(0));
    EXPECT_TRUE(handle.deviceGrid<BuildT>(0));
    EXPECT_FALSE(handle.deviceGrid<BuildT>(1));
    EXPECT_FALSE(handle.data());// no grid was yet allocated on the CPU

    //timer.start("Allocating and copying grid from GPU to CPU");
    auto *grid = handle.grid<BuildT>();// no grid on the CPU
    EXPECT_FALSE(grid);
    handle.deviceDownload();// creates a copy on the CPU
    EXPECT_TRUE(handle.deviceData());
    EXPECT_TRUE(handle.data());
    auto *data = handle.gridData();
    EXPECT_TRUE(data);
    grid = handle.grid<BuildT>();
    EXPECT_TRUE(grid);
    EXPECT_EQ(nanovdb::Vec3d(voxelSize), grid->voxelSize());
    EXPECT_EQ(pointCount, grid->pointCount());
    EXPECT_TRUE(nanovdb::CoordBBox::createCube(min, max-1).isInside(grid->indexBBox()));
    //std::cerr << grid->indexBBox() << std::endl;

    EXPECT_STREQ("World64: Vec3<double> point coordinates in world space", grid->blindMetaData(0).mName);
    {
        auto mgrHdl = nanovdb::createNodeManager(*grid);
        auto *mgr = mgrHdl.mgr<BuildT>();
        EXPECT_TRUE(mgr);
        for (uint32_t i=0; i<mgr->leafCount(); ++i) {
            const auto &leaf = mgr->leaf(i);
            for (int j=0; j<512; ++j) {
                EXPECT_LE(leaf.getValue(j), maxPointsPerLeaf);
                if (leaf.isActive(j)) {
                    if (j>0) {
                        EXPECT_LE(leaf.getValue(j) - leaf.getValue(j-1), maxPointsPerVoxel);
                    } else {
                        EXPECT_LE(leaf.getValue(0), maxPointsPerVoxel);
                    }
                } else if (j>0) {
                    EXPECT_EQ(leaf.getValue(j), leaf.getValue(j-1));
                } else {
                    EXPECT_EQ(leaf.getValue(0), 0u);
                }
            }// loop over voxels
        }// loop over leaf nodes
    }

    //timer.restart("Parallel unit-testing on CPU");
    nanovdb::forEach(points,[&](const nanovdb::Range1D &r){
        nanovdb::PointAccessor<Vec3T, BuildT> acc(*grid);
        EXPECT_TRUE(acc);
        const Vec3T *start = nullptr, *stop = nullptr;
        for (size_t i=r.begin(); i!=r.end(); ++i) {
            const nanovdb::Coord ijk = grid->worldToIndex(points[i]).round();
            EXPECT_TRUE(acc.probeLeaf(ijk)!=nullptr);
            EXPECT_TRUE(acc.isActive(ijk));
            EXPECT_LE(acc.getValue(ijk), pointCount);
            const auto *leaf = acc.get<nanovdb::GetLeaf<BuildT>>(ijk);
            EXPECT_TRUE(leaf);
            const auto offset = leaf->CoordToOffset(ijk);
            EXPECT_EQ(ijk, leaf->offsetToGlobalCoord(offset));
            const uint64_t count = acc.voxelPoints(ijk, start, stop);
            EXPECT_TRUE(start);
            EXPECT_TRUE(stop);
            EXPECT_LT(start, stop);
            EXPECT_LE(count, maxPointsPerVoxel);
            bool test = false;
            for (uint64_t j=0; test == false && j<count; ++j) {
                const nanovdb::Vec3d &xyz = start[j];
                test = nanovdb::isApproxZero<double>( (points[i] - xyz).lengthSqr() );
            }
            EXPECT_TRUE(test);
        }
    });

    //timer.stop();
}// Large_CudaPointsToGrid_World64

TEST(TestNanoVDBCUDA, Sphere_CudaPointsToGrid_World32)
{
    using BuildT = nanovdb::Points;
    using Vec3T  = nanovdb::Vec3f;

    nanovdb::CpuTimer timer("Generate sphere with points");
    auto pointsHandle = nanovdb::createPointSphere(8, 100.0, nanovdb::Vec3d(0.0), 0.5);
    timer.stop();

    auto *pointGrid = pointsHandle.grid<uint32_t>();
    EXPECT_TRUE(pointGrid);
    nanovdb::PointAccessor<Vec3T> acc2(*pointGrid);
    EXPECT_TRUE(acc2);
    const Vec3T *begin, *end;
    const size_t pointCount = acc2.gridPoints(begin, end);
    EXPECT_TRUE(begin);
    EXPECT_TRUE(end);
    EXPECT_LT(begin, end);

    const size_t pointSize = pointCount * sizeof(Vec3T);
    //std::cerr << "Point count = " << pointCount << ", point footprint: " << (pointSize >> 20) << " MB" << std::endl;
    //std::cerr << "Upper count: " << pointGrid->tree().nodeCount(2) << ", lower count: " << pointGrid->tree().nodeCount(1)
    //          << ", leaf count: " << pointGrid->tree().nodeCount(0) << ", voxelSize = " << pointGrid->voxelSize()[0] << std::endl;

    //timer.start("Allocating "+std::to_string(pointSize >> 20)+" MB on the GPU");
    Vec3T* d_points;
    cudaCheck(hipMalloc(&d_points, pointSize));
    //timer.restart("Copying points from CPU to GPU");
    cudaCheck(hipMemcpy(d_points, begin, pointSize, hipMemcpyHostToDevice));
    //timer.stop();

    timer.start("Building grid on GPU from "+std::to_string(pointCount)+" points");
    nanovdb::CudaPointsToGrid<BuildT> converter(pointGrid->map());
    //converter.setVerbose();
    auto handle = converter.getHandle(d_points, pointCount);
    timer.stop();
    cudaCheck(hipFree(d_points));
    //std::cerr << "Grid size: " << (handle.size() >> 20) << " MB" << std::endl;

    const uint32_t maxPointsPerVoxel = converter.maxPointsPerVoxel();
    const uint32_t maxPointsPerLeaf  = converter.maxPointsPerLeaf();
    EXPECT_GT(maxPointsPerVoxel, 0u);
    EXPECT_LT(maxPointsPerLeaf, 1024u);
    EXPECT_LE(maxPointsPerVoxel, maxPointsPerLeaf);
    //std::cerr << "maxPointsPerLeaf = " << maxPointsPerLeaf << " maxPointsPerVoxel = " << maxPointsPerVoxel << std::endl;

    EXPECT_TRUE(handle.deviceData());// grid only exists on the GPU
    EXPECT_TRUE(handle.deviceGrid<BuildT>());
    EXPECT_FALSE(handle.deviceGrid<int>(0));
    EXPECT_TRUE(handle.deviceGrid<BuildT>(0));
    EXPECT_FALSE(handle.deviceGrid<BuildT>(1));
    EXPECT_FALSE(handle.data());// no grid was yet allocated on the CPU

    //timer.start("Allocating and copying grid from GPU to CPU");
    auto *grid = handle.grid<BuildT>();// no grid on the CPU
    EXPECT_FALSE(grid);
    handle.deviceDownload();// creates a copy on the CPU
    EXPECT_TRUE(handle.deviceData());
    EXPECT_TRUE(handle.data());
    auto *data = handle.gridData();
    EXPECT_TRUE(data);
    grid = handle.grid<BuildT>();
    EXPECT_TRUE(grid);
    EXPECT_EQ(pointGrid->voxelSize(), grid->voxelSize());
    //std::cerr << grid->indexBBox() << std::endl;

    EXPECT_STREQ("World32: Vec3<float> point coordinates in world space", grid->blindMetaData(0).mName);

    {
        auto mgrHdl = nanovdb::createNodeManager(*grid);
        auto *mgr = mgrHdl.mgr<BuildT>();
        EXPECT_TRUE(mgr);
        for (uint32_t i=0; i<mgr->leafCount(); ++i) {
            const auto &leaf = mgr->leaf(i);
            for (int j=0; j<512; ++j) {
                EXPECT_LE(leaf.getValue(j), maxPointsPerLeaf);
                if (leaf.isActive(j)) {
                    if (j>0) {
                        EXPECT_LE(leaf.getValue(j) - leaf.getValue(j-1), maxPointsPerVoxel);
                    } else {
                        EXPECT_LE(leaf.getValue(0), maxPointsPerVoxel);
                    }
                } else if (j>0) {
                    EXPECT_EQ(leaf.getValue(j), leaf.getValue(j-1));
                } else {
                    EXPECT_EQ(leaf.getValue(0), 0u);
                }
            }// loop over voxels
        }// loop over leaf nodes
    }

    //timer.restart("Parallel unit-testing on CPU");
    nanovdb::forEach(0u, pointCount, 1u,[&](const nanovdb::Range1D &r){
        nanovdb::PointAccessor<Vec3T, BuildT> acc(*grid);
        EXPECT_TRUE(acc);
        const Vec3T *start = nullptr, *stop = nullptr;
        for (size_t i=r.begin(); i!=r.end(); ++i) {
            const nanovdb::Coord ijk = grid->worldToIndex(begin[i]).round();
            EXPECT_TRUE(acc.probeLeaf(ijk)!=nullptr);
            EXPECT_TRUE(acc.isActive(ijk));
            EXPECT_LE(acc.getValue(ijk), pointCount);
            const auto *leaf = acc.get<nanovdb::GetLeaf<BuildT>>(ijk);
            EXPECT_TRUE(leaf);
            const auto offset = leaf->CoordToOffset(ijk);
            EXPECT_EQ(ijk, leaf->offsetToGlobalCoord(offset));
            const uint64_t count = acc.voxelPoints(ijk, start, stop);
            EXPECT_TRUE(start);
            EXPECT_TRUE(stop);
            EXPECT_LT(start, stop);
            EXPECT_LE(count, maxPointsPerVoxel);
            bool test = false;
            for (uint64_t j=0; test == false && j<count; ++j) {
                const nanovdb::Vec3f &xyz = start[j];
                test = (begin[i] - xyz).length() < 1e-9;
            }
            EXPECT_TRUE(test);
        }
    });

    //timer.stop();
}// Sphere_CudaPointsToGrid_World32

TEST(TestNanoVDBCUDA, Sphere_CudaPointsToGrid_Voxel32)
{
    using BuildT = nanovdb::Points;
    using Vec3T  = nanovdb::Vec3f;

    nanovdb::CpuTimer timer("Generate sphere with points");
    auto pointsHandle = nanovdb::createPointSphere(8, 100.0, nanovdb::Vec3d(0.0), 0.5);
    timer.stop();

    auto *pointGrid = pointsHandle.grid<uint32_t>();
    EXPECT_TRUE(pointGrid);
    nanovdb::PointAccessor<Vec3T, uint32_t> acc2(*pointGrid);
    EXPECT_TRUE(acc2);
    const Vec3T *begin, *end;
    const size_t pointCount = acc2.gridPoints(begin, end);
    EXPECT_TRUE(begin);
    EXPECT_TRUE(end);
    EXPECT_LT(begin, end);

    const size_t pointSize = pointCount * sizeof(Vec3T);
    //std::cerr << "Point count = " << pointCount << ", point footprint: " << (pointSize >> 20) << " MB" << std::endl;
    //std::cerr << "Upper count: " << pointGrid->tree().nodeCount(2) << ", lower count: " << pointGrid->tree().nodeCount(1)
    //          << ", leaf count: " << pointGrid->tree().nodeCount(0) << ", voxelSize = " << pointGrid->voxelSize()[0] << std::endl;

    //timer.start("Allocating "+std::to_string(pointSize >> 20)+" MB on the GPU");
    Vec3T* d_points;
    cudaCheck(hipMalloc(&d_points, pointSize));
    //timer.restart("Copying points from CPU to GPU");
    cudaCheck(hipMemcpy(d_points, begin, pointSize, hipMemcpyHostToDevice));
    //timer.stop();

    timer.start("Building grid on GPU from "+std::to_string(pointCount)+" points");
    /////////////////////////////////////////////////////////////////////////
    nanovdb::CudaPointsToGrid<BuildT> converter(pointGrid->map());
    //converter.setVerbose();
    converter.setPointType(nanovdb::PointType::Voxel32);
    auto handle = converter.getHandle(d_points, pointCount);
    /////////////////////////////////////////////////////////////////////////
    timer.stop();
    cudaCheck(hipFree(d_points));
    //std::cerr << "Grid size: " << (handle.size() >> 20) << " MB" << std::endl;

    const uint32_t maxPointsPerVoxel = converter.maxPointsPerVoxel();
    const uint32_t maxPointsPerLeaf  = converter.maxPointsPerLeaf();
    EXPECT_GT(maxPointsPerVoxel, 0u);
    EXPECT_LT(maxPointsPerLeaf, 1024u);
    EXPECT_LE(maxPointsPerVoxel, maxPointsPerLeaf);
    //std::cerr << "maxPointsPerLeaf = " << maxPointsPerLeaf << " maxPointsPerVoxel = " << maxPointsPerVoxel << std::endl;

    EXPECT_TRUE(handle.deviceData());// grid only exists on the GPU
    EXPECT_TRUE(handle.deviceGrid<BuildT>());
    EXPECT_FALSE(handle.deviceGrid<int>(0));
    EXPECT_TRUE(handle.deviceGrid<BuildT>(0));
    EXPECT_FALSE(handle.deviceGrid<BuildT>(1));
    EXPECT_FALSE(handle.data());// no grid was yet allocated on the CPU

    //timer.start("Allocating and copying grid from GPU to CPU");
    auto *grid = handle.grid<BuildT>();// no grid on the CPU
    EXPECT_FALSE(grid);
    handle.deviceDownload();// creates a copy on the CPU
    EXPECT_TRUE(handle.deviceData());
    EXPECT_TRUE(handle.data());
    auto *data = handle.gridData();
    EXPECT_TRUE(data);
    grid = handle.grid<BuildT>();
    EXPECT_TRUE(grid);
    EXPECT_EQ(pointGrid->voxelSize(), grid->voxelSize());
    //std::cerr << grid->indexBBox() << std::endl;

    EXPECT_STREQ("Voxel32: Vec3<float> point coordinates in voxel space", grid->blindMetaData(0).mName);

    {
        auto mgrHdl = nanovdb::createNodeManager(*grid);
        auto *mgr = mgrHdl.mgr<BuildT>();
        EXPECT_TRUE(mgr);
        for (uint32_t i=0; i<mgr->leafCount(); ++i) {
            const auto &leaf = mgr->leaf(i);
            for (int j=0; j<512; ++j) {
                EXPECT_LE(leaf.getValue(j), maxPointsPerLeaf);
                if (leaf.isActive(j)) {
                    if (j>0) {
                        EXPECT_LE(leaf.getValue(j) - leaf.getValue(j-1), maxPointsPerVoxel);
                    } else {
                        EXPECT_LE(leaf.getValue(0), maxPointsPerVoxel);
                    }
                } else if (j>0) {
                    EXPECT_EQ(leaf.getValue(j), leaf.getValue(j-1));
                } else {
                    EXPECT_EQ(leaf.getValue(0), 0u);
                }
            }// loop over voxels
        }// loop over leaf nodes
    }

    //timer.restart("Parallel unit-testing on CPU");
    nanovdb::forEach(0u, pointCount, 1u,[&](const nanovdb::Range1D &r){
        nanovdb::PointAccessor<Vec3T, BuildT> acc(*grid);
        EXPECT_TRUE(acc);
        const Vec3T *start = nullptr, *stop = nullptr;
        for (size_t i=r.begin(); i!=r.end(); ++i) {
            const nanovdb::Coord ijk = grid->worldToIndex(begin[i]).round();
            EXPECT_TRUE(acc.probeLeaf(ijk)!=nullptr);
            EXPECT_TRUE(acc.isActive(ijk));
            EXPECT_LE(acc.getValue(ijk), pointCount);
            const auto *leaf = acc.get<nanovdb::GetLeaf<BuildT>>(ijk);
            EXPECT_TRUE(leaf);
            const auto offset = leaf->CoordToOffset(ijk);
            EXPECT_EQ(ijk, leaf->offsetToGlobalCoord(offset));
            const uint64_t count = acc.voxelPoints(ijk, start, stop);
            EXPECT_TRUE(start);
            EXPECT_TRUE(stop);
            EXPECT_LT(start, stop);
            EXPECT_LE(count, maxPointsPerVoxel);
            bool test = false;
            for (uint64_t j=0; test == false && j<count; ++j) {
                const nanovdb::Vec3f voxel = start[j];// local coordinates relative to voxel
                EXPECT_GE(voxel[0], -0.5f);
                EXPECT_GE(voxel[1], -0.5f);
                EXPECT_GE(voxel[2], -0.5f);
                EXPECT_LE(voxel[0],  0.5f);
                EXPECT_LE(voxel[1],  0.5f);
                EXPECT_LE(voxel[2],  0.5f);
                test = (begin[i] - nanovdb::voxelToWorld(voxel, ijk, grid->map())).length() < 1e-9;
            }
            EXPECT_TRUE(test);
        }
    });

    //timer.stop();
}// Sphere_CudaPointsToGrid_Voxel32

TEST(TestNanoVDBCUDA, Sphere_CudaPointsToGrid_Voxel16)
{
    EXPECT_EQ(6u, sizeof(nanovdb::Vec3u16));
    using BuildT = nanovdb::Points;
    using Vec3T  = nanovdb::Vec3f;

    nanovdb::CpuTimer timer("Generate sphere with points");
    auto pointsHandle = nanovdb::createPointSphere(8, 100.0, nanovdb::Vec3d(0.0), 0.5);
    timer.stop();

    auto *pointGrid = pointsHandle.grid<uint32_t>();
    EXPECT_TRUE(pointGrid);
    nanovdb::PointAccessor<Vec3T, uint32_t> acc2(*pointGrid);
    EXPECT_TRUE(acc2);
    const Vec3T *begin, *end;
    const size_t pointCount = acc2.gridPoints(begin, end);
    EXPECT_TRUE(begin);
    EXPECT_TRUE(end);
    EXPECT_LT(begin, end);

    const size_t pointSize = pointCount * sizeof(Vec3T);
    //std::cerr << "Point count = " << pointCount << ", point footprint: " << (pointSize >> 20) << " MB" << std::endl;
    //std::cerr << "Upper count: " << pointGrid->tree().nodeCount(2) << ", lower count: " << pointGrid->tree().nodeCount(1)
    //          << ", leaf count: " << pointGrid->tree().nodeCount(0) << ", voxelSize = " << pointGrid->voxelSize()[0] << std::endl;

    //timer.start("Allocating "+std::to_string(pointSize >> 20)+" MB on the GPU");
    Vec3T* d_points;
    cudaCheck(hipMalloc(&d_points, pointSize));
    //timer.restart("Copying points from CPU to GPU");
    cudaCheck(hipMemcpy(d_points, begin, pointSize, hipMemcpyHostToDevice));
    //timer.stop();

    timer.start("Building grid on GPU from "+std::to_string(pointCount)+" points");
    /////////////////////////////////////////////////////////////////////////
    nanovdb::CudaPointsToGrid<BuildT> converter(pointGrid->map());
    //converter.setVerbose();
    converter.setPointType(nanovdb::PointType::Voxel16);
    auto handle = converter.getHandle(d_points, pointCount);
    /////////////////////////////////////////////////////////////////////////
    timer.stop();
    cudaCheck(hipFree(d_points));
    //std::cerr << "Grid size: " << (handle.size() >> 20) << " MB" << std::endl;

    const uint32_t maxPointsPerVoxel = converter.maxPointsPerVoxel();
    const uint32_t maxPointsPerLeaf  = converter.maxPointsPerLeaf();
    EXPECT_GT(maxPointsPerVoxel, 0u);
    EXPECT_LT(maxPointsPerLeaf, 1024u);
    EXPECT_LE(maxPointsPerVoxel, maxPointsPerLeaf);
    //std::cerr << "maxPointsPerLeaf = " << maxPointsPerLeaf << " maxPointsPerVoxel = " << maxPointsPerVoxel << std::endl;

    EXPECT_TRUE(handle.deviceData());// grid only exists on the GPU
    EXPECT_TRUE(handle.deviceGrid<BuildT>());
    EXPECT_FALSE(handle.deviceGrid<int>(0));
    EXPECT_TRUE(handle.deviceGrid<BuildT>(0));
    EXPECT_FALSE(handle.deviceGrid<BuildT>(1));
    EXPECT_FALSE(handle.data());// no grid was yet allocated on the CPU

    //timer.start("Allocating and copying grid from GPU to CPU");
    auto *grid = handle.grid<BuildT>();// no grid on the CPU
    EXPECT_FALSE(grid);
    handle.deviceDownload();// creates a copy on the CPU
    EXPECT_TRUE(handle.deviceData());
    EXPECT_TRUE(handle.data());
    auto *data = handle.gridData();
    EXPECT_TRUE(data);
    grid = handle.grid<BuildT>();
    EXPECT_TRUE(grid);
    EXPECT_EQ(pointGrid->voxelSize(), grid->voxelSize());
    //std::cerr << grid->indexBBox() << std::endl;

    EXPECT_STREQ("Voxel16: Vec3<uint16_t> point coordinates in voxel space", grid->blindMetaData(0).mName);

    {
        auto mgrHdl = nanovdb::createNodeManager(*grid);
        auto *mgr = mgrHdl.mgr<BuildT>();
        EXPECT_TRUE(mgr);
        for (uint32_t i=0; i<mgr->leafCount(); ++i) {
            const auto &leaf = mgr->leaf(i);
            for (int j=0; j<512; ++j) {
                EXPECT_LE(leaf.getValue(j), maxPointsPerLeaf);
                if (leaf.isActive(j)) {
                    if (j>0) {
                        EXPECT_LE(leaf.getValue(j) - leaf.getValue(j-1), maxPointsPerVoxel);
                    } else {
                        EXPECT_LE(leaf.getValue(0), maxPointsPerVoxel);
                    }
                } else if (j>0) {
                    EXPECT_EQ(leaf.getValue(j), leaf.getValue(j-1));
                } else {
                    EXPECT_EQ(leaf.getValue(0), 0u);
                }
            }// loop over voxels
        }// loop over leaf nodes
    }

    //timer.restart("Parallel unit-testing on CPU");
    nanovdb::forEach(0u, pointCount, 1u,[&](const nanovdb::Range1D &r){
        nanovdb::PointAccessor<nanovdb::Vec3u16, BuildT> acc(*grid);
        EXPECT_TRUE(acc);
        const nanovdb::Vec3u16 *start = nullptr, *stop = nullptr;
        for (size_t i=r.begin(); i!=r.end(); ++i) {
            const nanovdb::Coord ijk = grid->worldToIndex(begin[i]).round();
            EXPECT_TRUE(acc.probeLeaf(ijk)!=nullptr);
            EXPECT_TRUE(acc.isActive(ijk));
            EXPECT_LE(acc.getValue(ijk), pointCount);
            const auto *leaf = acc.get<nanovdb::GetLeaf<BuildT>>(ijk);
            EXPECT_TRUE(leaf);
            const auto offset = leaf->CoordToOffset(ijk);
            EXPECT_EQ(ijk, leaf->offsetToGlobalCoord(offset));
            const uint64_t count = acc.voxelPoints(ijk, start, stop);
            EXPECT_TRUE(start);
            EXPECT_TRUE(stop);
            EXPECT_LT(start, stop);
            EXPECT_LE(count, maxPointsPerVoxel);
            bool test = false;
            for (uint64_t j=0; test == false && j<count; ++j) {
                test = (begin[i] - nanovdb::voxelToWorld(start[j], ijk, grid->map())).length() < 1e-6;
            }
        }
    });

    //timer.stop();
}// Sphere_CudaPointsToGrid_Voxel16

TEST(TestNanoVDBCUDA, Sphere_CudaPointsToGrid_Voxel8)
{
    EXPECT_EQ(3u, sizeof(nanovdb::Vec3u8));

    using BuildT = nanovdb::Points;
    using Vec3T  = nanovdb::Vec3f;

    nanovdb::CpuTimer timer("Generate sphere with points");
    auto pointsHandle = nanovdb::createPointSphere(8, 100.0, nanovdb::Vec3d(0.0), 0.5);
    timer.stop();

    auto *pointGrid = pointsHandle.grid<uint32_t>();
    EXPECT_TRUE(pointGrid);
    std::cerr << "nanovdb::bbox = " << pointGrid->indexBBox() << " voxel count = " << pointGrid->activeVoxelCount() << std::endl;
    nanovdb::PointAccessor<Vec3T, uint32_t> acc2(*pointGrid);
    EXPECT_TRUE(acc2);
    const Vec3T *begin, *end;
    const size_t pointCount = acc2.gridPoints(begin, end);
    EXPECT_TRUE(begin);
    EXPECT_TRUE(end);
    EXPECT_LT(begin, end);

    const size_t pointSize = pointCount * sizeof(Vec3T);
    //std::cerr << "Point count = " << pointCount << ", point footprint: " << (pointSize >> 20) << " MB" << std::endl;
    //std::cerr << "Upper count: " << pointGrid->tree().nodeCount(2) << ", lower count: " << pointGrid->tree().nodeCount(1)
    //          << ", leaf count: " << pointGrid->tree().nodeCount(0) << ", voxelSize = " << pointGrid->voxelSize()[0] << std::endl;

    //timer.start("Allocating "+std::to_string(pointSize >> 20)+" MB on the GPU");
    Vec3T* d_points;
    cudaCheck(hipMalloc(&d_points, pointSize));
    //timer.restart("Copying points from CPU to GPU");
    cudaCheck(hipMemcpy(d_points, begin, pointSize, hipMemcpyHostToDevice));
    //timer.stop();

    timer.start("Building grid on GPU from "+std::to_string(pointCount)+" points");
    /////////////////////////////////////////////////////////////////////////
    //auto handle = nanovdb::cudaPointsToGrid(d_points, pointCount, nanovdb::PointType::Voxel8);
    nanovdb::CudaPointsToGrid<BuildT> converter(pointGrid->map());
    //converter.setVerbose();
    converter.setPointType(nanovdb::PointType::Voxel8);
    auto handle = converter.getHandle(d_points, pointCount);
    /////////////////////////////////////////////////////////////////////////
    timer.stop();
    cudaCheck(hipFree(d_points));
    //std::cerr << "Grid size: " << (handle.size() >> 20) << " MB" << std::endl;

    const uint32_t maxPointsPerVoxel = converter.maxPointsPerVoxel();
    const uint32_t maxPointsPerLeaf  = converter.maxPointsPerLeaf();
    EXPECT_GT(maxPointsPerVoxel, 0u);
    EXPECT_LT(maxPointsPerLeaf, 1024u);
    EXPECT_LE(maxPointsPerVoxel, maxPointsPerLeaf);
    //std::cerr << "maxPointsPerLeaf = " << maxPointsPerLeaf << " maxPointsPerVoxel = " << maxPointsPerVoxel << std::endl;

    EXPECT_TRUE(handle.deviceData());// grid only exists on the GPU
    EXPECT_TRUE(handle.deviceGrid<BuildT>());
    EXPECT_FALSE(handle.deviceGrid<int>(0));
    EXPECT_TRUE(handle.deviceGrid<BuildT>(0));
    EXPECT_FALSE(handle.deviceGrid<BuildT>(1));
    EXPECT_FALSE(handle.data());// no grid was yet allocated on the CPU

    //timer.start("Allocating and copying grid from GPU to CPU");
    auto *grid = handle.grid<BuildT>();// no grid on the CPU
    EXPECT_FALSE(grid);
    handle.deviceDownload();// creates a copy on the CPU
    EXPECT_TRUE(handle.deviceData());
    EXPECT_TRUE(handle.data());
    auto *data = handle.gridData();
    EXPECT_TRUE(data);
    grid = handle.grid<BuildT>();
    EXPECT_TRUE(grid);
    EXPECT_EQ(pointGrid->voxelSize(), grid->voxelSize());
    std::cerr << grid->indexBBox() << std::endl;

    EXPECT_STREQ("Voxel8: Vec3<uint8_t> point coordinates in voxel space", grid->blindMetaData(0).mName);

    {
        auto mgrHdl = nanovdb::createNodeManager(*grid);
        auto *mgr = mgrHdl.mgr<BuildT>();
        EXPECT_TRUE(mgr);
        for (uint32_t i=0; i<mgr->leafCount(); ++i) {
            const auto &leaf = mgr->leaf(i);
            for (int j=0; j<512; ++j) {
                EXPECT_LE(leaf.getValue(j), maxPointsPerLeaf);
                if (leaf.isActive(j)) {
                    if (j>0) {
                        EXPECT_LE(leaf.getValue(j) - leaf.getValue(j-1), maxPointsPerVoxel);
                    } else {
                        EXPECT_LE(leaf.getValue(0), maxPointsPerVoxel);
                    }
                } else if (j>0) {
                    EXPECT_EQ(leaf.getValue(j), leaf.getValue(j-1));
                } else {
                    EXPECT_EQ(leaf.getValue(0), 0u);
                }
            }// loop over voxels
        }// loop over leaf nodes
    }

    //timer.restart("Parallel unit-testing on CPU");
    nanovdb::forEach(0u, pointCount, 1u,[&](const nanovdb::Range1D &r){
        nanovdb::PointAccessor<nanovdb::Vec3u8, BuildT> acc(*grid);
        EXPECT_TRUE(acc);
        const nanovdb::Vec3u8 *start = nullptr, *stop = nullptr;
        for (size_t i=r.begin(); i!=r.end(); ++i) {
            const nanovdb::Coord ijk = grid->worldToIndex(begin[i]).round();
            EXPECT_TRUE(acc.probeLeaf(ijk)!=nullptr);
            EXPECT_TRUE(acc.isActive(ijk));
            EXPECT_LE(acc.getValue(ijk), pointCount);
            const auto *leaf = acc.get<nanovdb::GetLeaf<BuildT>>(ijk);
            EXPECT_TRUE(leaf);
            const auto offset = leaf->CoordToOffset(ijk);
            EXPECT_EQ(ijk, leaf->offsetToGlobalCoord(offset));
            const uint64_t count = acc.voxelPoints(ijk, start, stop);
            EXPECT_TRUE(start);
            EXPECT_TRUE(stop);
            EXPECT_LT(start, stop);
            EXPECT_LE(count, maxPointsPerVoxel);
            bool test = false;
            for (uint64_t j=0; test == false && j<count; ++j) {
                test = (begin[i] - nanovdb::voxelToWorld(start[j], ijk, grid->map())).length() < 1e-2;
            }
            EXPECT_TRUE(test);
        }
    });
    //timer.stop();
}// Sphere_CudaPointsToGrid_Voxel8

TEST(TestNanoVDBCUDA, Sphere_CudaPointsToGrid_PointID)
{
    EXPECT_EQ(3u, sizeof(nanovdb::Vec3u8));

    using BuildT = nanovdb::Points;
    using Vec3T  = nanovdb::Vec3f;

    nanovdb::CpuTimer timer("Generate sphere with points");
    auto pointsHandle = nanovdb::createPointSphere(8, 100.0, nanovdb::Vec3d(0.0), 0.5);
    timer.stop();

    auto *pointGrid = pointsHandle.grid<uint32_t>();
    EXPECT_TRUE(pointGrid);
    std::cerr << "nanovdb::bbox = " << pointGrid->indexBBox() << " voxel count = " << pointGrid->activeVoxelCount() << std::endl;
    nanovdb::PointAccessor<Vec3T, uint32_t> acc2(*pointGrid);
    EXPECT_TRUE(acc2);
    const Vec3T *begin, *end;
    const size_t pointCount = acc2.gridPoints(begin, end);
    EXPECT_TRUE(begin);
    EXPECT_TRUE(end);
    EXPECT_LT(begin, end);

    const size_t pointSize = pointCount * sizeof(Vec3T);
    //std::cerr << "Point count = " << pointCount << ", point footprint: " << (pointSize >> 20) << " MB" << std::endl;
    //std::cerr << "Upper count: " << pointGrid->tree().nodeCount(2) << ", lower count: " << pointGrid->tree().nodeCount(1)
    //          << ", leaf count: " << pointGrid->tree().nodeCount(0) << ", voxelSize = " << pointGrid->voxelSize()[0] << std::endl;

    //timer.start("Allocating "+std::to_string(pointSize >> 20)+" MB on the GPU");
    Vec3T* d_points;
    cudaCheck(hipMalloc(&d_points, pointSize));
    //timer.restart("Copying points from CPU to GPU");
    cudaCheck(hipMemcpy(d_points, begin, pointSize, hipMemcpyHostToDevice));
    //timer.stop();

    timer.start("Building grid on GPU from "+std::to_string(pointCount)+" points");
    /////////////////////////////////////////////////////////////////////////
    //auto handle = nanovdb::cudaPointsToGrid(d_points, pointCount, nanovdb::PointType::Voxel8);
    nanovdb::CudaPointsToGrid<BuildT> converter(pointGrid->map());
    converter.setVerbose(2);
    converter.setPointType(nanovdb::PointType::PointID);
    auto handle = converter.getHandle(d_points, pointCount);
    /////////////////////////////////////////////////////////////////////////
    timer.stop();
    cudaCheck(hipFree(d_points));
    //std::cerr << "Grid size: " << (handle.size() >> 20) << " MB" << std::endl;

    const uint32_t maxPointsPerVoxel = converter.maxPointsPerVoxel();
    const uint32_t maxPointsPerLeaf  = converter.maxPointsPerLeaf();
    EXPECT_GT(maxPointsPerVoxel, 0u);
    EXPECT_LT(maxPointsPerLeaf, 1024u);
    EXPECT_LE(maxPointsPerVoxel, maxPointsPerLeaf);
    //std::cerr << "maxPointsPerLeaf = " << maxPointsPerLeaf << " maxPointsPerVoxel = " << maxPointsPerVoxel << std::endl;

    EXPECT_TRUE(handle.deviceData());// grid only exists on the GPU
    EXPECT_TRUE(handle.deviceGrid<BuildT>());
    EXPECT_FALSE(handle.deviceGrid<int>(0));
    EXPECT_TRUE(handle.deviceGrid<BuildT>(0));
    EXPECT_FALSE(handle.deviceGrid<BuildT>(1));
    EXPECT_FALSE(handle.data());// no grid was yet allocated on the CPU

    //timer.start("Allocating and copying grid from GPU to CPU");
    auto *grid = handle.grid<BuildT>();// no grid on the CPU
    EXPECT_FALSE(grid);
    handle.deviceDownload();// creates a copy on the CPU
    EXPECT_TRUE(handle.deviceData());
    EXPECT_TRUE(handle.data());
    auto *data = handle.gridData();
    EXPECT_TRUE(data);
    grid = handle.grid<BuildT>();
    EXPECT_TRUE(grid);
    EXPECT_EQ(pointGrid->voxelSize(), grid->voxelSize());
    //std::cerr << grid->indexBBox() << std::endl;

    EXPECT_STREQ("PointID: uint32_t indices to points", grid->blindMetaData(0).mName);

    {
        auto mgrHdl = nanovdb::createNodeManager(*grid);
        auto *mgr = mgrHdl.mgr<BuildT>();
        EXPECT_TRUE(mgr);
        for (uint32_t i=0; i<mgr->leafCount(); ++i) {
            const auto &leaf = mgr->leaf(i);
            for (int j=0; j<512; ++j) {
                EXPECT_LE(leaf.getValue(j), maxPointsPerLeaf);
                if (leaf.isActive(j)) {
                    if (j>0) {
                        EXPECT_LE(leaf.getValue(j) - leaf.getValue(j-1), maxPointsPerVoxel);
                    } else {
                        EXPECT_LE(leaf.getValue(0), maxPointsPerVoxel);
                    }
                } else if (j>0) {
                    EXPECT_EQ(leaf.getValue(j), leaf.getValue(j-1));
                } else {
                    EXPECT_EQ(leaf.getValue(0), 0u);
                }
            }// loop over voxels
        }// loop over leaf nodes
    }

    //timer.restart("Parallel unit-testing on CPU");
    nanovdb::forEach(0u, pointCount, 1u,[&](const nanovdb::Range1D &r){
        nanovdb::PointAccessor<uint32_t, BuildT> acc(*grid);
        EXPECT_TRUE(acc);
        const uint32_t *start = nullptr, *stop = nullptr;
        for (size_t i=r.begin(); i!=r.end(); ++i) {
            const nanovdb::Coord ijk = grid->worldToIndex(begin[i]).round();
            EXPECT_TRUE(acc.probeLeaf(ijk)!=nullptr);
            EXPECT_TRUE(acc.isActive(ijk));
            EXPECT_LE(acc.getValue(ijk), pointCount);
            const auto *leaf = acc.get<nanovdb::GetLeaf<BuildT>>(ijk);
            EXPECT_TRUE(leaf);
            const auto offset = leaf->CoordToOffset(ijk);
            EXPECT_EQ(ijk, leaf->offsetToGlobalCoord(offset));
            const uint64_t count = acc.voxelPoints(ijk, start, stop);
            EXPECT_TRUE(start);
            EXPECT_TRUE(stop);
            EXPECT_LT(start, stop);
            EXPECT_LE(count, maxPointsPerVoxel);
        }
    });

    //timer.stop();
}// Sphere_CudaPointsToGrid_PointID

TEST(TestNanoVDBCUDA, NanoGrid_Rgba8)
{
    using BuildT = nanovdb::Rgba8;
    using GridT = nanovdb::NanoGrid<BuildT>;
    const size_t num_points = 1;
    nanovdb::Coord coords[num_points] = {nanovdb::Coord(1, 2, 3)}, *d_coords = nullptr;
    cudaCheck(hipMalloc(&d_coords, num_points * sizeof(nanovdb::Coord)));
    cudaCheck(hipMemcpy(d_coords, coords, num_points * sizeof(nanovdb::Coord), hipMemcpyHostToDevice));// CPU -> GPU

    nanovdb::CudaPointsToGrid<BuildT> converter;
    auto handle = converter.getHandle(d_coords, num_points);
    cudaCheck(hipFree(d_coords));

    EXPECT_TRUE(handle.deviceData());// grid only exists on the GPU
    EXPECT_FALSE(handle.data());// no grid was yet allocated on the CPU

    const uint64_t size = sizeof(GridT) +
                          sizeof(GridT::TreeType) +
                          GridT::RootType::memUsage(1) +
                          sizeof(GridT::UpperNodeType) +
                          sizeof(GridT::LowerNodeType) +
                          sizeof(GridT::LeafNodeType);
    EXPECT_EQ(handle.size(), size);

    GridT *grid = handle.grid<BuildT>();// no grid on the CPU
    EXPECT_FALSE(grid);
    handle.deviceDownload();// creates a copy up the CPU
    EXPECT_TRUE(handle.deviceData());
    EXPECT_TRUE(handle.data());
    auto *data = handle.gridData();
    EXPECT_TRUE(data);
    grid = handle.grid<BuildT>();
    EXPECT_TRUE(grid);
}// NanoGrid_Rgba8

TEST(TestNanoVDBCUDA, cudaAddBlindData)
{
    using BuildT = float;
    using GridT  = nanovdb::NanoGrid<BuildT>;
    const size_t num_points = 2;
    nanovdb::Coord coords[num_points] = {nanovdb::Coord(1, 2, 3), nanovdb::Coord(10,20,8)}, *d_coords = nullptr;
    cudaCheck(hipMalloc(&d_coords, num_points * sizeof(nanovdb::Coord)));
    cudaCheck(hipMemcpy(d_coords, coords, num_points * sizeof(nanovdb::Coord), hipMemcpyHostToDevice));// CPU -> GPU
    auto handle = nanovdb::cudaVoxelsToGrid<BuildT>(d_coords, num_points);
    cudaCheck(hipFree(d_coords));
    EXPECT_TRUE(handle.deviceData());// grid only exists on the GPU
    EXPECT_FALSE(handle.data());// no grid was yet allocated on the CPU
    const uint64_t size = sizeof(GridT) +
                          sizeof(GridT::TreeType) +
                          GridT::RootType::memUsage(1) +
                          sizeof(GridT::UpperNodeType) +
                          sizeof(GridT::LowerNodeType) +
                          2*sizeof(GridT::LeafNodeType);
    EXPECT_EQ(handle.size(), size);
    GridT *d_grid = handle.deviceGrid<BuildT>();// no grid on the CPU
    EXPECT_TRUE(d_grid);
    float *d_blind = nullptr, blind[num_points] = {1.2f, 3.0f};
    cudaCheck(hipMalloc(&d_blind, num_points * sizeof(float)));
    cudaCheck(hipMemcpy(d_blind, blind, num_points * sizeof(float), hipMemcpyHostToDevice));// CPU -> GPU

    nanovdb::GpuTimer timer("cudaAddBlindData");
    auto handle2 = nanovdb::cudaAddBlindData(d_grid, d_blind, num_points);
    cudaCheck(hipFree(d_blind));
    timer.stop();
    EXPECT_TRUE(handle2.deviceData());// grid only exists on the GPU
    EXPECT_FALSE(handle2.data());// no grid was yet allocated on the CPU
    EXPECT_EQ(handle2.size(), handle.size() + sizeof(nanovdb::GridBlindMetaData) + nanovdb::AlignUp<NANOVDB_DATA_ALIGNMENT>(num_points*sizeof(float)));

    auto *grid2 = handle2.grid<BuildT>();// no grid on the CPU
    EXPECT_FALSE(grid2);
    handle2.deviceDownload();// creates a copy on the CPU
    EXPECT_TRUE(handle2.deviceData());
    EXPECT_TRUE(handle2.data());
    auto *data = handle2.gridData();
    EXPECT_TRUE(data);
    grid2 = handle2.grid<BuildT>();
    EXPECT_TRUE(grid2);
    EXPECT_EQ(nanovdb::Vec3d(1.0), grid2->voxelSize());
    EXPECT_EQ(1u, grid2->blindDataCount());
    const auto &bd2 = grid2->blindMetaData(0);
    EXPECT_EQ(num_points, bd2.mValueCount);
    EXPECT_EQ(nanovdb::GridBlindDataSemantic::Unknown, bd2.mSemantic);
    EXPECT_EQ(nanovdb::GridBlindDataClass::Unknown, bd2.mDataClass);
    EXPECT_EQ(nanovdb::GridType::Float, bd2.mDataType);
    EXPECT_STREQ("", bd2.mName);
    const float *dataPtr = bd2.getBlindData<float>();
    EXPECT_TRUE(dataPtr);
    for (size_t i=0; i<num_points; ++i) EXPECT_EQ(blind[i], dataPtr[i]);

    GridT *d_grid2 = handle2.deviceGrid<BuildT>();// no grid on the CPU
    EXPECT_TRUE(d_grid2);

    nanovdb::Vec3f *d_blind2 = nullptr, blind2[num_points] = {nanovdb::Vec3f(1.2f), nanovdb::Vec3f(3.0f)};
    cudaCheck(hipMalloc(&d_blind2, num_points * sizeof(nanovdb::Vec3f)));
    cudaCheck(hipMemcpy(d_blind2, blind2, num_points * sizeof(nanovdb::Vec3f), hipMemcpyHostToDevice));// CPU -> GPU

    auto handle3 = nanovdb::cudaAddBlindData(d_grid2, d_blind2, num_points,
                                             nanovdb::GridBlindDataClass::AttributeArray,
                                             nanovdb::GridBlindDataSemantic::PointPosition,
                                             "this is a test");
    cudaCheck(hipFree(d_blind2));
    handle3.deviceDownload();// creates a copy on the CPU
    GridT *grid3 = handle3.grid<BuildT>();// no grid on the CPU
    EXPECT_TRUE(grid3);
    EXPECT_EQ(2, grid3->blindDataCount());

    const auto &bd3 = grid3->blindMetaData(0);
    EXPECT_EQ(num_points, bd3.mValueCount);
    EXPECT_EQ(nanovdb::GridBlindDataSemantic::Unknown, bd3.mSemantic);
    EXPECT_EQ(nanovdb::GridBlindDataClass::Unknown, bd3.mDataClass);
    EXPECT_EQ(nanovdb::GridType::Float, bd3.mDataType);
    EXPECT_STREQ("", bd3.mName);
    dataPtr = grid3->getBlindData<float>(0);
    EXPECT_TRUE(dataPtr);
    for (size_t i=0; i<num_points; ++i) EXPECT_EQ(blind[i], dataPtr[i]);

    const auto &bd4 = grid3->blindMetaData(1);
    EXPECT_EQ(num_points, bd4.mValueCount);
    EXPECT_EQ(nanovdb::GridBlindDataSemantic::PointPosition, bd4.mSemantic);
    EXPECT_EQ(nanovdb::GridBlindDataClass::AttributeArray, bd4.mDataClass);
    EXPECT_EQ(nanovdb::GridType::Vec3f, bd4.mDataType);
    EXPECT_STREQ("this is a test", bd4.mName);
    auto *dataPtr2 = grid3->getBlindData<nanovdb::Vec3f>(1);
    EXPECT_TRUE(dataPtr2);
    for (size_t i=0; i<num_points; ++i) EXPECT_EQ(blind2[i], dataPtr2[i]);
}// cudaAddBlindData
